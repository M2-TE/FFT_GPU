    #define N 1024
    #define M 2*N
    #define D N  
       __global__  void fft(float* A , float* ROT)
       {
    __shared__ float SA[M],SB[M],SROT[N];
    short j   = threadIdx.x;
    short n = logf(N)/logf(2);
    SROT[j] = ROT[j];
   
    SA[j] = A[j];
    SA[j+blockDim.x] = A[j+blockDim.x];
    __syncthreads();
   

    short i = j/2;
    short k = j%2;
   
    short ind0 = i<<1;
    short ind1 = (i<<1) + N;
    short ind2 = i<<2;
    short sign =(-2)*k+1;


    for(short s= 1; s<= n ; s++)
      {
        SB[ind2+k] = SA[ind0+k] + SA[ind1+k];
        SB[ind2+2+k] = SA[ind0+k] - SA[ind1+k];
        short r0 = (i/(1<<(s-1)))*(1<<(s-1));
        SA[ind2+k] = SB[ind2+k];
        SA[ind2+2+k] = sign*SB[ind2+2]*SROT[2*r0+k] + SB[ind2+3]*SROT[2*r0+(!k)];
	__syncthreads();
      }
    

        A[j] = SA[j];
        A[j+blockDim.x] = SA[j+blockDim.x];
       

      }

       
#include <hip/hip_runtime.h>
#include  <stdio.h>
       #include  <math.h>
       int  main()
       {
       
       float A[2*N];
       float *Ad;
       float ROT[N];
       float *ROTd;

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
     
       int memsize= 2*N * sizeof(float);
       int rotsize = N* sizeof(float);

        for(int i=0; i<N; i++)
        {
            A[2*i]  = i;
            A[2*i+1]= i;   
        }
        for(int j=0; j < (N/2); j++)
        {
            ROT[2*j]= cosf((j*(6.2857))/N);
            ROT[2*j+1]=sinf((j*(6.2857))/N);   
        }

       hipMalloc((void**)&Ad, memsize);
       hipMalloc((void**)&ROTd, rotsize);

       hipMemcpy(Ad, A, memsize,  hipMemcpyHostToDevice);
      

           // __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
           dim3 gridDim(1,1);
           dim3 blockDim(D,1);

	hipEventRecord(start, 0);
	 hipMemcpy(ROTd, ROT, rotsize,  hipMemcpyHostToDevice);
       fft<<<gridDim , blockDim>>>(Ad,ROTd );
	 hipEventRecord(stop, 0);
	   hipEventSynchronize(stop);

       hipMemcpy(A, Ad, memsize,  hipMemcpyDeviceToHost);
       hipMemcpy(ROT, ROTd, rotsize,  hipMemcpyDeviceToHost);

            printf("The  outputs are: \n");
            /*for (int l=0; l< N; l++)
            printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ",2*l,A[2*l],2*l+1,A[2*l+1]);*/
  	 hipEventElapsedTime(&time, start, stop);
       	    printf ("Time for the kernel: %f us\n", time*1000.0);


     }
