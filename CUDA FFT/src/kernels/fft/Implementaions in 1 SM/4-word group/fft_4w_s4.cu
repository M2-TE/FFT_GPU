	#define N 256
	#define M 2*N
	#define D N   
       __global__  void fft(float* A , float* ROT ) 
       { 
	__shared__ float SA[M],SB[M],SROT[N];
	short j   = threadIdx.x;
	short n = logf(N)/logf(4);
	SROT[j] = ROT[j];
	
	SA[j] = A[j];
	SA[j+blockDim.x] = A[j+blockDim.x];
	__syncthreads();
	
	short ind0,r0,r1,sign1,sign2,index1,index2;
	short i = j/2;
	short k = j%2;
	short h = i/2;
	short l = j%4;
	short m = i%2;
		
	for(short s=1; s<=n; s++)
	{
		short p = M/(1<<(2*s));
		short ind_tmp = 2*(h+(h/(1<<2*(n-s)))*(1<<2*(n-s))*3);
		short r_tmp = (h%(1<<2*(n-s)))*(1<<2*(s-1));
				
		ind0 = ind_tmp+l*p;
		
		//stage1:
		sign1= m*(-2)+1;
		r0 = r_tmp + k*(N/4);
		index1 = ind0+sign1*2*p;
				
		SB[ind0] = SA[index1] + sign1*SA[ind0];
		SB[ind0+1] = SA[index1+1] + sign1*SA[ind0+1];
		
		short inx1 = ind0 - m*2*p + m;
		short inx2 = ind0 + (!m)*2*p ; 

		SA[inx1]   = SB[inx1];
		SA[inx2+m] = sign1*SB[inx2]*SROT[2*r0+m] + SB[inx2+1]*SROT[2*r0+(!m)];
		__syncthreads();
		//stage2:
		
		sign2 = k*(-2)+1;
		r1 = r_tmp*2; 
		index2 = ind0+sign2*p;
		
		SB[ind0] = SA[index2] + sign2*SA[ind0];
		SB[ind0+1] = SA[index2+1] + sign2*SA[ind0+1];
			
		short inx3 = ind0 - k*p + k;
		short inx4 = ind0 + (!k)*p ; 
	
		SA[inx3] = SB[inx3];
		SA[inx4+k] = sign2*SB[inx4]*SROT[2*r1+k] + SB[inx4+1]*SROT[2*r1+(!k)];

		/*if(m==0)
		{
			SA[ind0]   = SB[ind0];
			SA[index1] = SB[index1]*SROT[2*r0] + SB[index1+1]*SROT[2*r0+1];
		}
		else
		{
			SA[index1+1] = SB[index1+1];
			SA[ind0+1]   = -SB[ind0]*SROT[2*r0+1] + SB[ind0+1]*SROT[2*r0];
		}*/
	__syncthreads();
	}
	
	A[j] = SA[j];
	A[j+blockDim.x] = SA[j+blockDim.x];

    }

       
#include <hip/hip_runtime.h>
#include  <stdio.h> 
       #include  <math.h>
       int  main() 
       { 
        
           float A[2*N]; 
           float *Ad;
	   float ROT[N];
	   float *ROTd; 
	  
           int memsize= 2*N * sizeof(float); 
	   int rotsize = N* sizeof(float);

		for(int i=0; i<N; i++)
		{
			A[2*i]  = i;
			A[2*i+1]= i;	
		}
		for(int j=0; j < (N/2); j++)
		{
			
			ROT[2*j]= cosf((j*(6.2857))/N);
			ROT[2*j+1]=sinf((j*(6.2857))/N);	
		}

           hipMalloc((void**)&Ad, memsize);
	   hipMalloc((void**)&ROTd, rotsize); 

           hipMemcpy(Ad, A, memsize,  hipMemcpyHostToDevice); 
	   hipMemcpy(ROTd, ROT, rotsize,  hipMemcpyHostToDevice); 

           //__global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter); 
           dim3 gridDim(1,1);
           dim3 blockDim(D,1);
	   fft<<<gridDim , blockDim>>>(Ad,ROTd );
           hipMemcpy(A, Ad, memsize,  hipMemcpyDeviceToHost); 
	   hipMemcpy(ROT, ROTd, rotsize,  hipMemcpyDeviceToHost);

            printf("The  outputs are: \n");
            for (int l=0; l< N; l++) 
            printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ",2*l,A[2*l],2*l+1,A[2*l+1]); 

     }
	
	
		
