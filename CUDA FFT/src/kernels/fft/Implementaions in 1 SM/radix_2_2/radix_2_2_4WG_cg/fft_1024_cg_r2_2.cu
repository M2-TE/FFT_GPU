    #define N 1024
    #define M 2*N
    #define D N  
    #define n 10
   __global__  void fft(float* A , float* ROT)
    {
    __shared__ float SA[M],SB[M],SROT[M];
    short j   = threadIdx.x;
   // short n = logf(N)/logf(2);
    SROT[j] = ROT[j];
    SROT[j+blockDim.x] = ROT[j+blockDim.x];
   
    SA[j] = A[j];
    SA[j+blockDim.x] = A[j+blockDim.x];
    __syncthreads();
   

    short i = j>>1;
    short k = j%2;
   
    short ind0 = i<<1;
    short ind1 = (i<<1) + N;
    short ind2 = j<<1;
    short signk =-(k<<1) + 1;
    short s;

    //stage1:
	
	short r0 = k*(j>>(n-1));
	short signr0 = -(r0<<1) + 1;
	SB[ind2 + r0] = signr0*(SA[ind0] + signk*SA[ind1]);
	SB[ind2 + (!r0)] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();

    //stage2:
	s=2;
	
	short r1 = (((j%2)<<1) + ((j>>1)%2))*(j>>s);
	
	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];
	__syncthreads();
	SB[ind2] = SA[ind2]*SROT[r1<<1] + SA[ind2+1]*SROT[(r1<<1)+1];
	SB[ind2+1] = -SA[ind2]*SROT[(r1<<1)+1] + SA[ind2+1]*SROT[r1<<1];
         //SA[j]=SROT[j];
	 //SA[j+blockDim.x] =SROT[j+blockDim.x];
	 __syncthreads();
    //stage3:
	SA[ind2 + r0] = signr0*(SB[ind0] + signk*SB[ind1]);
	SA[ind2 + (!r0)] = SB[ind0+1] + signk*SB[ind1+1];
	__syncthreads();
    //stage4:
	s=4;
	short r3 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>s);
	SB[ind2] = SA[ind0] + signk*SA[ind1];
	SB[ind2+1] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();
	SA[ind2] = SB[ind2]*SROT[r3<<1] + SB[ind2+1]*SROT[(r3<<1)+1];
	SA[ind2+1] = -SB[ind2]*SROT[(r3<<1)+1] + SB[ind2+1]*SROT[r3<<1];
	 __syncthreads();
     //stage5:
	SB[ind2 + r0] = signr0*(SA[ind0] + signk*SA[ind1]);
	SB[ind2 + (!r0)] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();
    //stage6:
	s=6;
	short r5 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>s);
	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];
	__syncthreads();
	SB[ind2] = SA[ind2]*SROT[r5<<1] + SA[ind2+1]*SROT[(r5<<1)+1];
	SB[ind2+1] = -SA[ind2]*SROT[(r5<<1)+1] + SA[ind2+1]*SROT[r5<<1];
	 __syncthreads();
   //stage7:
	SA[ind2 + r0] = signr0*(SB[ind0] + signk*SB[ind1]);
	SA[ind2 + (!r0)] = SB[ind0+1] + signk*SB[ind1+1];

	__syncthreads();
    //stage8:
	s=8;
	short r7 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>s);
	SB[ind2] = SA[ind0] + signk*SA[ind1];
	SB[ind2+1] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();
	SA[ind2] = SB[ind2]*SROT[r7<<1] + SB[ind2+1]*SROT[(r7<<1)+1];
	SA[ind2+1] = -SB[ind2]*SROT[(r7<<1)+1] + SB[ind2+1]*SROT[r7<<1];

	 __syncthreads();
    //stage9:
	SB[ind2 + r0] = signr0*(SA[ind0] + signk*SA[ind1]);
	SB[ind2 + (!r0)] = SA[ind0+1] + signk*SA[ind1+1];

	__syncthreads();
    //stage10:
	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];
	
	__syncthreads();
		A[j] = SA[j];
		A[j+blockDim.x] = SA[j+blockDim.x];
     
    }

         
#include <hip/hip_runtime.h>
#include  <stdio.h> 
       #include  <math.h>
       int  main() 
       { 
        
           float A[2*N]; 
           float *Ad;
	   float ROT[2*N];
	   float *ROTd; 
	  	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
           int memsize= 2*N * sizeof(float); 
	   int rotsize = 2*N* sizeof(float);


		for(int i=0; i<N; i++)
		{
			A[2*i]  = i;
			A[2*i+1]= i;	
		}
		for(int j=0; j < (N); j++)
		{
			
			ROT[2*j]= cosf((j*(6.2857))/N);
			ROT[2*j+1]=sinf((j*(6.2857))/N);	
		}

           hipMalloc((void**)&Ad, memsize);
	   hipMalloc((void**)&ROTd, rotsize); 

           hipMemcpy(Ad, A, memsize,  hipMemcpyHostToDevice); 
	  

           //__global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter); 
           dim3 gridDim(1,1);
           dim3 blockDim(D,1);

	   hipEventRecord(start, 0);
	    hipMemcpy(ROTd, ROT, rotsize,  hipMemcpyHostToDevice); 
	   fft<<<gridDim , blockDim>>>(Ad,ROTd );
	   hipEventRecord(stop, 0);
	   hipEventSynchronize(stop);

           hipMemcpy(A, Ad, memsize,  hipMemcpyDeviceToHost); 
	   hipMemcpy(ROT, ROTd, rotsize,  hipMemcpyDeviceToHost);

           /* printf("The  outputs are: \n");
            for (int l=0; l< N; l++) 
            printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ",2*l,A[2*l],2*l+1,A[2*l+1]); */

	    	hipEventElapsedTime(&time, start, stop);
       	    printf ("Time for the kernel: %f us\n", time*1000.0);

     }


