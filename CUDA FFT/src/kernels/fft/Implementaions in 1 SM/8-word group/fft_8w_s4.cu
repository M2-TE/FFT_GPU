	#define N 512
	#define M 2*N
	#define D N/2    
       __global__  void fft(float* A , float* ROT) 
       { 
	__shared__ float SA[M],SB[M],SROT[N];
	short j   = threadIdx.x;
	short n = logf(N)/logf(8);

	SROT[j] = ROT[j];
	SROT[j+blockDim.x] = ROT[j+blockDim.x];
	
	SA[j] = A[j];
	SA[j+blockDim.x] = A[j+blockDim.x];
	SA[j+2*blockDim.x] = A[j+2*blockDim.x];
	SA[j+3*blockDim.x] = A[j+3*blockDim.x];
	
	__syncthreads();
	
	short ind0, ind1,r0,r1,r2,ind_tmp,r_tmp;
	short i = j/2;
	short k = i%2;
	short h = i/2;
	short l = j%4;
	short m = j%2;
	for(short s=1; s<=n; s++)
	{
		short p = M/(1<<(3*s));
		ind_tmp = 2*(h+(h/(1<<3*(n-s)))*(1<<3*(n-s))*7);
		ind0 = ind_tmp+p*l;
		ind1 = ind_tmp+4*p+p*l;
		r_tmp = (h%(1<<3*(n-s)))*(1<<3*(s-1));
		
		//1st stage:
		r0 = r_tmp + l*(N/8); 
		SB[ind0]   =  SA[ind0] + SA[ind1];
		SB[ind0+1] =  SA[ind0+1] + SA[ind1+1]; 
		SB[ind1]   =  SA[ind0] - SA[ind1];
		SB[ind1+1] =  SA[ind0+1] - SA[ind1+1];

		SA[ind0]   =  SB[ind0];
		SA[ind0+1] =  SB[ind0+1];
		SA[ind1]   =  SB[ind1]*SROT[2*r0] + SB[ind1+1]*SROT[2*r0+1];
		SA[ind1+1] =  -SB[ind1]*SROT[2*r0+1] + SB[ind1+1]*SROT[2*r0]; 
		//__syncthreads();

		//2nd stage:
		short tmp_s2 = ind0 + 2*p*k;
		r1 = 2*r_tmp + m*(N/4);
		
		SB[tmp_s2]   =  SA[tmp_s2] + SA[tmp_s2+2*p];
		SB[tmp_s2+1] =  SA[tmp_s2+1] + SA[tmp_s2+2*p+1]; 
		SB[tmp_s2+2*p]   =  SA[tmp_s2] - SA[tmp_s2+2*p];
		SB[tmp_s2+2*p+1] =  SA[tmp_s2+1] - SA[tmp_s2+2*p+1];

		SA[tmp_s2]   =  SB[tmp_s2];
		SA[tmp_s2+1] =  SB[tmp_s2+1];
		SA[tmp_s2+2*p]   =  SB[tmp_s2+2*p]*SROT[2*r1] + SB[tmp_s2+2*p+1]*SROT[2*r1+1];
		SA[tmp_s2+2*p+1] =  -SB[tmp_s2+2*p]*SROT[2*r1+1] + SB[tmp_s2+2*p+1]*SROT[2*r1];
		//__syncthreads();

		//3rd stage:
		short tmp_s3 = ind0+l*p;
		r2 = 4*r_tmp;
		
		SB[tmp_s3]   =  SA[tmp_s3] + SA[tmp_s3+p];
		SB[tmp_s3+1] =  SA[tmp_s3+1] + SA[tmp_s3+p+1]; 
		SB[tmp_s3+p]   =  SA[tmp_s3] - SA[tmp_s3+p];
		SB[tmp_s3+p+1] =  SA[tmp_s3+1] - SA[tmp_s3+p+1];

		SA[tmp_s3]   =  SB[tmp_s3];
		SA[tmp_s3+1] =  SB[tmp_s3+1];
		SA[tmp_s3+p]   =  SB[tmp_s3+p]*SROT[2*r2] + SB[tmp_s3+p+1]*SROT[2*r2+1];
		SA[tmp_s3+p+1] =  -SB[tmp_s3+p]*SROT[2*r2+1] + SB[tmp_s3+p+1]*SROT[2*r2]; 
		
	}
		A[j] = SA[j];
		A[j+blockDim.x] = SA[j+blockDim.x];
		A[j+2*blockDim.x] = SA[j+2*blockDim.x];
		A[j+3*blockDim.x] = SA[j+3*blockDim.x];
      }

       
#include <hip/hip_runtime.h>
#include  <stdio.h> 
       #include  <math.h>
       int  main() 
       { 
        
           float A[2*N]; 
           float *Ad;
	   float ROT[N];
	   float *ROTd; 
	  
           int memsize= 2*N * sizeof(float); 
	   int rotsize = N* sizeof(float);

		for(int i=0; i<N; i++)
		{
			A[2*i]  = i;
			A[2*i+1]= i;	
		}
		for(int j=0; j < (N/2); j++)
		{
			
			ROT[2*j]= cosf((j*(6.2857))/N);
			ROT[2*j+1]=sinf((j*(6.2857))/N);	
		}

           hipMalloc((void**)&Ad, memsize);
	   hipMalloc((void**)&ROTd, rotsize); 

           hipMemcpy(Ad, A, memsize,  hipMemcpyHostToDevice); 
	   hipMemcpy(ROTd, ROT, rotsize,  hipMemcpyHostToDevice); 

           // __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter); 
           dim3 gridDim(1,1);
           dim3 blockDim(D,1);
	   fft<<<gridDim , blockDim>>>(Ad,ROTd );
           hipMemcpy(A, Ad, memsize,  hipMemcpyDeviceToHost); 
	   hipMemcpy(ROT, ROTd, rotsize,  hipMemcpyDeviceToHost);

            printf("The  outputs are: \n");
            for (int l=0; l< N; l++) 
            printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ",2*l,A[2*l],2*l+1,A[2*l+1]); 

     }

		




		
		
