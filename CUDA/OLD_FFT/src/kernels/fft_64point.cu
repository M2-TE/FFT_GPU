
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_math_constants.h>

typedef unsigned int uint;
#define THREADS_PER_CHUNK 8 // should be 8
#define CHUNK_SIZE 64 // should be 64
#define INPUT_SIZE 64 // number of complex values

#define INDEXING_ALIASES const uint idx = threadIdx.x; const uint idy = threadIdx.y
#define STEPPING_ALIASES const uint xStep = wordSize * 2; const uint yStep = fftSize * 2
#define TEMPLATE_A template <uint fftSize>
#define TEMPLATE_B template <uint inputShuffleSize = 0, uint outputShuffleSize = 0>

// utils
__device__ void debug_values(float* S)
{
	const uint wordSize = 4;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	for (uint i = 0; i < 8; i++) {
		uint index = idx * xStep + i * 2;
		printf("[Thread %d Value %d]\tReal: %f\t\tImag: %f\n", idx, i, S[index], S[index + 1]);
	}
}
__device__ void mem_transfer(float* src, float* dst)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// TODO: make this read/write data in coalescence
	uint index = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i++) {
		dst[index + i] = src[index + i];
	}
}
// shuffling variants (deprecated)
__device__ void shuffle(float* S)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// need to store values in temp array before writing
	// (not all threads write all their 8 values at once -> undefined behaviour otherwise)
	float temps[wordSize * 2];
	uint offsetSrc = idx * xStep; // could abstain from storing this in register, its only usage will be in offsetSrc + i, which is a single multiply+add operation
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// shuffle index bits (b6, b5, b4) <-> (b3, b2, b1) + (b0)
		uint index = offsetSrc + i;
		uint upper = index & 0b111'000'0;
		uint lower = index & 0b000'111'0;
		index = (upper >> 3) | (lower << 3);
		index += idy * yStep;

		// write both real and imag parts to temp
		temps[i]     = S[index];
		temps[i + 1] = S[index + 1];
	}

	// then write values using temp array
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {
		uint index = offsetDst + i;
		S[index]     = temps[i];
		S[index + 1] = temps[i + 1];
	}
}
__device__ void shuffleB(float* S)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	float temp[wordSize * 2];
	uint offsetSrc = idx * 2 + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// read value at target (shuffled) index
		uint iSrc = i * wordSize + offsetSrc;
		temp[i] = S[iSrc];
		temp[i + 1] = S[iSrc + 1];

		//S[iDst]	  = S[iSrc];
		//S[iDst + 1] = S[iSrc + 1];
	}
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// write value to src (pre-shuffle) index
		uint iDst = i + offsetDst;
		S[iDst] = temp[i];
		S[iDst + 1] = temp[i + 1];
	}
}
// rotations and stuff
TEMPLATE_A __device__ void rotate(float* S)
{
	const uint wordSize = 8;
	INDEXING_ALIASES;
	STEPPING_ALIASES;
	const float scaling = 2 * HIP_PI_F / fftSize;

	for (uint i = 0; i < wordSize; i++) {

		float a = (float)idx;	// floor(index / 8) tid is basically that, no need for more calculations
		float b = (float)i; 	// mod(i, 8) i is already guarenteed to be between 0 and 8
		float phi = a * b;
		float ang = scaling * phi;
		float c = cosf(ang);
		float s = sinf(ang);

		uint index = idx * xStep + idy * yStep + i * 2;
		float real = S[index];
		float imag = S[index + 1];
		S[index]	 = c * real + s * imag;
		S[index + 1] = c * imag - s * real;
	}
}
TEMPLATE_B __device__ void execute_8point_fft_shuffled(float* S)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;

	// stage 1
	{
		// butterflies
		if constexpr (inputShuffleSize) {
			const uint offsetR = idx * 2 + idy * yStep;
			const uint offsetI = offsetR + 1;
			x0 =  S[inputShuffleSize *  0 + offsetR] + S[inputShuffleSize *  8 + offsetR]; // R
			x1 =  S[inputShuffleSize *  0 + offsetI] + S[inputShuffleSize *  8 + offsetI]; // I
			x8 =  S[inputShuffleSize *  0 + offsetR] - S[inputShuffleSize *  8 + offsetR]; // R
			x9 =  S[inputShuffleSize *  0 + offsetI] - S[inputShuffleSize *  8 + offsetI]; // I

			x2 =  S[inputShuffleSize *  2 + offsetR] + S[inputShuffleSize * 10 + offsetR]; // R
			x3 =  S[inputShuffleSize *  2 + offsetI] + S[inputShuffleSize * 10 + offsetI]; // I
			x10 = S[inputShuffleSize *  2 + offsetR] - S[inputShuffleSize * 10 + offsetR]; // R
			x11 = S[inputShuffleSize *  2 + offsetI] - S[inputShuffleSize * 10 + offsetI]; // I

			x4 =  S[inputShuffleSize *  4 + offsetR] + S[inputShuffleSize * 12 + offsetR]; // R
			x5 =  S[inputShuffleSize *  4 + offsetI] + S[inputShuffleSize * 12 + offsetI]; // I
			x12 = S[inputShuffleSize *  4 + offsetI] - S[inputShuffleSize * 12 + offsetI]; // R (swapped)
			x13 = S[inputShuffleSize * 12 + offsetR] - S[inputShuffleSize *  4 + offsetR]; // I (swapped)

			x6 =  S[inputShuffleSize *  6 + offsetR] + S[inputShuffleSize * 14 + offsetR]; // R
			x7 =  S[inputShuffleSize *  6 + offsetI] + S[inputShuffleSize * 14 + offsetI]; // I
			x14 = S[inputShuffleSize *  6 + offsetR] - S[inputShuffleSize * 14 + offsetR]; // R
			x15 = S[inputShuffleSize *  6 + offsetI] - S[inputShuffleSize * 14 + offsetI]; // I
		}
		else {
			uint index = idx * xStep + idy * yStep;
			x0 =  S[index +  0] + S[index +  8]; // R
			x1 =  S[index +  1] + S[index +  9]; // I
			x8 =  S[index +  0] - S[index +  8]; // R
			x9 =  S[index +  1] - S[index +  9]; // I
				  			 
			x2 =  S[index +  2] + S[index + 10]; // R
			x3 =  S[index +  3] + S[index + 11]; // I
			x10 = S[index +  2] - S[index + 10]; // R
			x11 = S[index +  3] - S[index + 11]; // I
							 
			x4 =  S[index +  4] + S[index + 12]; // R
			x5 =  S[index +  5] + S[index + 13]; // I
			x12 = S[index +  5] - S[index + 13]; // R (swapped)
			x13 = S[index + 12] - S[index +  4]; // I (swapped)

			x6 =  S[index +  6] + S[index + 14]; // R
			x7 =  S[index +  7] + S[index + 15]; // I
			x14 = S[index +  6] - S[index + 14]; // R
			x15 = S[index +  7] - S[index + 15]; // I
		}

		// rotations
		{
			const float coef = sqrtf(2.0f) / 2.0f;

			x10 = x10 * coef;
			x11 = x11 * coef;

			float temp = x10 + x11;
			x11 = x11 - x10;
			x10 = temp;


			x14 = x14 * coef;
			x15 = x15 * coef;

			temp = -x14 + x15;
			x15 = -x15 - x14;
			x14 = temp;
		}
	}

	// stage 2
	{
		// butterflies
		float tempR;
		float tempI;
		tempR = x0 + x4; // R
		tempI = x1 + x5; // I
		x4 = x0 - x4; // R
		x5 = x1 - x5; // I
		x0 = tempR;
		x1 = tempI;

		tempR = x2 + x6; // R
		tempI = x3 + x7; // I
		float tempR2 = x3 - x7;
		x7 = x6 - x2; // I (swapped)
		x6 = tempR2; // R (swapped)
		x2 = tempR;
		x3 = tempI;

		tempR = x8 + x12; // R
		tempI = x9 + x13; // I
		x12 = x8 - x12; // R
		x13 = x9 - x13; // I
		x8 = tempR;
		x9 = tempI;

		tempR = x10 + x14; // R
		tempI = x11 + x15; // I
		tempR2 = x11 - x15;
		x15 = x14 - x10; // I (swapped)
		x14 = tempR2; // R (swapped)
		x10 = tempR;
		x11 = tempI;
	}

	// stage 3
	{
		// butterflies (with bit reversal)
		if constexpr (outputShuffleSize) {
			const uint offsetR = idx * 2 + idy * yStep;
			const uint offsetI = offsetR + 1;
			
			S[outputShuffleSize *  0 + offsetR] =  x0 +  x2;
			S[outputShuffleSize *  0 + offsetI] =  x1 +  x3;
			S[outputShuffleSize *  2 + offsetR] =  x8 + x10;
			S[outputShuffleSize *  2 + offsetI] =  x9 + x11;

			S[outputShuffleSize *  4 + offsetR] =  x4 +  x6;
			S[outputShuffleSize *  4 + offsetI] =  x5 +  x7;
			S[outputShuffleSize *  6 + offsetR] = x12 + x14;
			S[outputShuffleSize *  6 + offsetI] = x13 + x15;

			S[outputShuffleSize *  8 + offsetR] =  x0 -  x2;
			S[outputShuffleSize *  8 + offsetI] =  x1 -  x3;
			S[outputShuffleSize * 10 + offsetR] =  x8 - x10;
			S[outputShuffleSize * 10 + offsetI] =  x9 - x11;

			S[outputShuffleSize * 12 + offsetR] =  x4 -  x6;
			S[outputShuffleSize * 12 + offsetI] =  x5 -  x7;
			S[outputShuffleSize * 14 + offsetR] = x12 - x14;
			S[outputShuffleSize * 14 + offsetI] = x13 - x15;
		}
		else {
			uint index = idx * xStep + idy * yStep;
			S[index +  0] =  x0 +  x2;
			S[index +  1] =  x1 +  x3;
			S[index +  2] =  x8 + x10;
			S[index +  3] =  x9 + x11;
					   
			S[index +  4] =  x4 +  x6;
			S[index +  5] =  x5 +  x7;
			S[index +  6] = x12 + x14;
			S[index +  7] = x13 + x15;
					   
			S[index +  8] =  x0 -  x2;
			S[index +  9] =  x1 -  x3;
			S[index + 10] =  x8 - x10;
			S[index + 11] =  x9 - x11;

			S[index + 12] =  x4 -  x6;
			S[index + 13] =  x5 -  x7;
			S[index + 14] = x12 - x14;
			S[index + 15] = x13 - x15;
		}
	}
}
TEMPLATE_B __device__ void execute_4point_fft_shuffled(float* S)
{
	const uint wordSize = 4;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7;

	// stage 1
	// butterflies + rotations
	if constexpr (inputShuffleSize) {
		
		const uint offsetR = idx * 2 + idy * yStep;
		const uint offsetI = offsetR + 1;
		x0 = S[inputShuffleSize * 0 + offsetR] + S[inputShuffleSize * 4 + offsetR]; // R
		x1 = S[inputShuffleSize * 0 + offsetI] + S[inputShuffleSize * 4 + offsetI]; // I
		x4 = S[inputShuffleSize * 0 + offsetR] - S[inputShuffleSize * 4 + offsetR]; // R
		x5 = S[inputShuffleSize * 0 + offsetI] - S[inputShuffleSize * 4 + offsetI]; // I

		x2 = S[inputShuffleSize * 2 + offsetR] + S[inputShuffleSize * 6 + offsetR]; // R
		x3 = S[inputShuffleSize * 2 + offsetI] + S[inputShuffleSize * 6 + offsetI]; // I
		x6 = S[inputShuffleSize * 2 + offsetI] - S[inputShuffleSize * 6 + offsetI]; // R (swapped)
		x7 = S[inputShuffleSize * 6 + offsetR] - S[inputShuffleSize * 2 + offsetR]; // I (swapped)
	}
	else {
		const uint index = idx * xStep + idy * yStep;
		x0 = S[index + 0] + S[index + 4]; // R
		x1 = S[index + 1] + S[index + 5]; // I
		x4 = S[index + 0] - S[index + 4]; // R
		x5 = S[index + 1] - S[index + 5]; // I

		x2 = S[index + 2] + S[index + 6]; // R
		x3 = S[index + 3] + S[index + 7]; // I
		x6 = S[index + 3] - S[index + 7]; // R (swapped)
		x7 = S[index + 6] - S[index + 2]; // I (swapped)
	}

	// stage 2
	// butterflies + bit reversal
	if constexpr (outputShuffleSize) {

		const uint offsetR = idx * 2 + idy * yStep;
		const uint offsetI = offsetR + 1;
		S[outputShuffleSize *  0 + offsetR] = x0 + x2;
		S[outputShuffleSize *  0 + offsetI] = x1 + x3;
		S[outputShuffleSize *  4 + offsetR] = x4 + x6;
		S[outputShuffleSize *  4 + offsetI] = x5 + x7;

		S[outputShuffleSize *  8 + offsetR] = x0 - x2;
		S[outputShuffleSize *  8 + offsetI] = x1 - x3;
		S[outputShuffleSize * 12 + offsetR] = x4 - x6;
		S[outputShuffleSize * 12 + offsetI] = x5 - x7;
	}
	else {
		const uint index = idx * xStep + idy * yStep;
		S[index + 0] = x0 + x2;
		S[index + 1] = x1 + x3;
		S[index + 2] = x4 + x6;
		S[index + 3] = x5 + x7;

		S[index + 4] = x0 - x2;
		S[index + 5] = x1 - x3;
		S[index + 6] = x4 - x6;
		S[index + 7] = x5 - x7;
	}
}
TEMPLATE_B __device__ void execute_2point_fft_shuffled(float* S)
{
	const uint wordSize = 2;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3;

	// stage 1
	// butterflies
	if constexpr (inputShuffleSize) {
		const uint offsetR = idx * 2 + idy * yStep;
		const uint offsetI = offsetR + 1;

		x0 = S[inputShuffleSize * 0 + offsetR] + S[inputShuffleSize * 2 + offsetR]; // R
		x1 = S[inputShuffleSize * 0 + offsetI] + S[inputShuffleSize * 2 + offsetI]; // I

		x2 = S[inputShuffleSize * 0 + offsetR] - S[inputShuffleSize * 2 + offsetR]; // R
		x3 = S[inputShuffleSize * 0 + offsetI] - S[inputShuffleSize * 2 + offsetI]; // I
	}
	else {
		const uint index = idx * xStep + idy * yStep;
		x0 = S[index + 0] + S[index + 2]; // R
		x1 = S[index + 1] + S[index + 3]; // I

		x2 = S[index + 0] - S[index + 2]; // R
		x3 = S[index + 1] - S[index + 3]; // I
	}

	// output only
	if constexpr (outputShuffleSize) {
		const uint offsetR = idx * 2 + idy * yStep;
		const uint offsetI = offsetR + 1;

		S[outputShuffleSize * 0 + offsetR] = x0;
		S[outputShuffleSize * 0 + offsetI] = x1;

		S[outputShuffleSize * 2 + offsetR] = x2;
		S[outputShuffleSize * 2 + offsetI] = x3;
	}
	else {
		S[idx * xStep + 0] = x0;
		S[idx * xStep + 1] = x1;
		S[idx * xStep + 2] = x2;
		S[idx * xStep + 3] = x3;
	}
}

__device__ void shuffle_A(float* S)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// need to store values in temp array before writing
	// (not all threads write all their 8 values at once -> undefined behaviour otherwise)
	float temps[wordSize * 2];
	uint offsetSrc = idx * xStep; // could abstain from storing this in register, its only usage will be in offsetSrc + i, which is a single multiply+add operation
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// shuffle index bits (b6, b5, b4) <-> (b3, b2, b1) + (b0)
		uint index = offsetSrc + i;
		uint upper = index & 0b1'000'0;
		uint lower = index & 0b0'111'0;
		index = (upper >> 3) | (lower << 1);
		index += idy * yStep;

		// write both real and imag parts to temp
		temps[i] = S[index];
		temps[i + 1] = S[index + 1];
	}

	// then write values using temp array
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {
		uint index = offsetDst + i;
		S[index] = temps[i];
		S[index + 1] = temps[i + 1];
	}
}
__device__ void shuffle_B(float* S)
{
	const uint wordSize = 8;
	const uint fftSize = 64;
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// need to store values in temp array before writing
	// (not all threads write all their 8 values at once -> undefined behaviour otherwise)
	float temps[wordSize * 2];
	uint offsetSrc = idx * xStep; // could abstain from storing this in register, its only usage will be in offsetSrc + i, which is a single multiply+add operation
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// shuffle index bits (b6, b5, b4) <-> (b3, b2, b1) + (b0)
		uint index = offsetSrc + i;
		uint upper = index & 0b111'0'0;
		uint lower = index & 0b000'1'0;
		index = (upper >> 1) | (lower << 3);
		index += idy * yStep;

		// write both real and imag parts to temp
		temps[i] = S[index];
		temps[i + 1] = S[index + 1];
	}

	// then write values using temp array
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {
		uint index = offsetDst + i;
		S[index] = temps[i];
		S[index + 1] = temps[i + 1];
	}
}

// core kernel
__global__ void fft(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);

	// input shuffle + first 8-point fft
	execute_8point_fft_shuffled<8, false>(S);

	// single rotation for each value
	rotate<64>(S);

	// input shuffle + second 8-point fft + output shuffle
	execute_8point_fft_shuffled<8, 8>(S);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}
__global__ void fft_32(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);

	// input shuffle + first 8-point fft
	execute_8point_fft_shuffled<4, false>(S);
	
	// single rotation for each value
	rotate<32>(S);

	// input shuffle + second fft (2x 4-point) + output shuffle
	// NOTE: shuffle size of 4 is normal shuffle, 8 is inverse
	execute_4point_fft_shuffled<8, 4>(S + 0);
	execute_4point_fft_shuffled<8, 4>(S + 8);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}
__global__ void fft_16(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);

	// input shuffle + first 8-point fft
	execute_8point_fft_shuffled<2, false>(S);

	// single rotation for each value
	rotate<16>(S);

	// input shuffle + second fft (2x 4-point) + output shuffle
	execute_2point_fft_shuffled<8, 2>(S + 0);
	execute_2point_fft_shuffled<8, 2>(S + 8);
	execute_2point_fft_shuffled<8, 2>(S + 16);
	execute_2point_fft_shuffled<8, 2>(S + 24);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}
__global__ void fft_old(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);

	// input shuffle
	shuffleB(S);
	// executing first 8-point FFT
	execute_8point_fft_shuffled<false, false>(S);

	// rotation + shuffle
	rotate<64>(S);
	shuffleB(S);

	// executing second 8-point FFT
	execute_8point_fft_shuffled<false, false>(S);

	// output shuffle
	shuffleB(S);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}

int main()
{
	float* pIN;
	float IN[2 * INPUT_SIZE];
	float OUT[2 * INPUT_SIZE];

	for (int i = 0; i < INPUT_SIZE; i++)
	{
		// DEBUGGING for advanced indexing
		IN[2 * i] = i % 64;
		IN[2 * i + 1] = i % 64;
	}

	int memsize = 2 * INPUT_SIZE * sizeof(float);
	hipMalloc((void**)&pIN, memsize);
	hipMemcpy(pIN, IN, memsize, hipMemcpyHostToDevice);

	// Gets rid of false flags with IntelliSense
#ifdef __HIPCC__
	#define KERNEL_GRID(grid, block) <<< grid, block >>>
#else
	#define KERNEL_GRID(grid, block)
#endif
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(THREADS_PER_CHUNK, INPUT_SIZE / CHUNK_SIZE, 1);
	fft KERNEL_GRID(gridDim, blockDim)(pIN, pIN);
	//fft_32 KERNEL_GRID(gridDim, blockDim)(pIN, pIN);
	//fft_16 KERNEL_GRID(gridDim, blockDim)(pIN, pIN);

	hipMemcpy(OUT, pIN, memsize, hipMemcpyDeviceToHost);
	printf("The  outputs are: \n");
	for (int l = 0; l < INPUT_SIZE; l++) {
		printf("RE:A[%d]=%10.2f\t\t\t, IM: A[%d]=%10.2f\t\t\t \n ", 2 * l, OUT[2 * l], 2 * l + 1, OUT[2 * l + 1]);
	}
}
