
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_math_constants.h>

typedef unsigned int uint;
#define INPUT_SIZE 64 // number of complex values
#define WORD_SIZE 8
#define TEMPLATE_A template <uint wordSize = WORD_SIZE, uint fftSize = 64>
#define TEMPLATE_B template <uint wordSize = WORD_SIZE, uint fftSize = 64, bool bShuffleInput = false, bool bShuffleOutput = false>
#define INDEXING_ALIASES const uint idx = threadIdx.x; const uint idy = threadIdx.y
#define STEPPING_ALIASES const uint xStep = wordSize * 2; const uint yStep = fftSize * 2

/// deprecated atm
__device__ void execute_8point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const float coef = sqrtf(2.0f) / 2.0f;
	const unsigned int wordSize = 8;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 8]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 9]; // I
	float x8 = IN[tid * step + 0] - IN[tid * step + 8]; // R
	float x9 = IN[tid * step + 1] - IN[tid * step + 9]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 10]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 11]; // I
	float x10 = IN[tid * step + 2] - IN[tid * step + 10]; // R
	float x11 = IN[tid * step + 3] - IN[tid * step + 11]; // I

	float x4 = IN[tid * step + 4] + IN[tid * step + 12]; // R
	float x5 = IN[tid * step + 5] + IN[tid * step + 13]; // I
	float x12 = IN[tid * step + 5] - IN[tid * step + 13]; // R (swapped)
	float x13 = IN[tid * step + 12] - IN[tid * step + 4]; // I (swapped)

	float x6 = IN[tid * step + 6] + IN[tid * step + 14]; // R
	float x7 = IN[tid * step + 7] + IN[tid * step + 15]; // I
	float x14 = IN[tid * step + 6] - IN[tid * step + 14]; // R
	float x15 = IN[tid * step + 7] - IN[tid * step + 15]; // I

	// rotations
	x10 = x10 * coef;
	x11 = x11 * coef;

	float temp = x10 + x11;
	x11 = x11 - x10;
	x10 = temp;


	x14 = x14 * coef;
	x15 = x15 * coef;

	temp = -x14 + x15;
	x15 = -x15 - x14;
	x14 = temp;

	// stage 2
	// butterflies
	float tempR;
	float tempI;
	tempR = x0 + x4; // R
	tempI = x1 + x5; // I
	x4 = x0 - x4; // R
	x5 = x1 - x5; // I
	x0 = tempR;
	x1 = tempI;

	tempR = x2 + x6; // R
	tempI = x3 + x7; // I
	float tempR2 = x3 - x7;
	x7 = x6 - x2; // I (swapped)
	x6 = tempR2; // R (swapped)
	x2 = tempR;
	x3 = tempI;

	tempR = x8 + x12; // R
	tempI = x9 + x13; // I
	x12 = x8 - x12; // R
	x13 = x9 - x13; // I
	x8 = tempR;
	x9 = tempI;

	tempR = x10 + x14; // R
	tempI = x11 + x15; // I
	tempR2 = x11 - x15;
	x15 = x14 - x10; // I (swapped)
	x14 = tempR2; // R (swapped)
	x10 = tempR;
	x11 = tempI;

	// stage 3
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x8 + x10;
	IN[tid * step + 3] = x9 + x11;

	IN[tid * step + 4] = x4 + x6;
	IN[tid * step + 5] = x5 + x7;
	IN[tid * step + 6] = x12 + x14;
	IN[tid * step + 7] = x13 + x15;

	IN[tid * step + 8] = x0 - x2;
	IN[tid * step + 9] = x1 - x3;
	IN[tid * step + 10] = x8 - x10;
	IN[tid * step + 11] = x9 - x11;

	IN[tid * step + 12] = x4 - x6;
	IN[tid * step + 13] = x5 - x7;
	IN[tid * step + 14] = x12 - x14;
	IN[tid * step + 15] = x13 - x15;
}
__device__ void execute_4point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 4;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 4]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 5]; // I
	float x4 = IN[tid * step + 0] - IN[tid * step + 4]; // R
	float x5 = IN[tid * step + 1] - IN[tid * step + 5]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 6]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 7]; // I
	float x6 = IN[tid * step + 3] - IN[tid * step + 7]; // R (swapped)
	float x7 = IN[tid * step + 6] - IN[tid * step + 2]; // I (swapped)

	// stage 2
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x4 + x6;
	IN[tid * step + 3] = x5 + x7;

	IN[tid * step + 4] = x0 - x2;
	IN[tid * step + 5] = x1 - x3;
	IN[tid * step + 6] = x4 - x6;
	IN[tid * step + 7] = x5 - x7;
}
__device__ void execute_2point_fft_deprecated(float* IN)
{
	//__shared__ float S[4];
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 2;
	const unsigned int step = wordSize * 2;

	// only really need to store 0 and 1 in registers
	// but compiler puts em all in registers anyways 
	// and this is just more readable:
	float x0 = IN[tid * step + 0];
	float x1 = IN[tid * step + 1];
	float x2 = IN[tid * step + 2];
	float x3 = IN[tid * step + 3];

	// stage 1
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x0 - x2;
	IN[tid * step + 3] = x1 - x3;
}
///

// utils
__device__ void debug_values(float* S)
{
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const uint tid = threadIdx.x;

	for (uint i = 0; i < 8; i++) {
		printf("[Thread %d Value %d]\tReal: %f\t\tImag: %f\n", tid, i, S[tid * step + i * 2], S[tid * step + i * 2 + 1]);
	}
}
TEMPLATE_A __device__ void mem_transfer(float* src, float* dst)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// TODO: make this read/write data in coalescence
	uint index = idy * yStep + idx * xStep;
	for (uint i = 0; i < wordSize * 2; i++) {
		dst[index + i] = src[index + i];
	}
}
// shuffling variants
TEMPLATE_A __device__ void shuffle(float* S)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	// need to store values in temp array before writing
	// (not all threads write all their 8 values at once -> undefined behaviour otherwise)
	float temps[wordSize * 2];
	uint offsetSrc = idx * xStep; // could abstain from storing this in register, its only usage will be in offsetSrc + i, which is a single multiply+add operation
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// shuffle index bits (b6, b5, b4) <-> (b3, b2, b1) + (b0)
		uint index = offsetSrc + i;
		uint upper = index & 0b111'000'0;
		uint lower = index & 0b000'111'0;
		index = (upper >> 3) | (lower << 3);
		index += idy * yStep;

		// write both real and imag parts to temp
		temps[i]     = S[index];
		temps[i + 1] = S[index + 1];
	}

	// then write values using temp array
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {
		uint index = offsetDst + i;
		S[index]     = temps[i];
		S[index + 1] = temps[i + 1];
	}
}
TEMPLATE_A __device__ void shuffleB(float* S)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	float temp[wordSize * 2];
	uint offsetSrc = idx * 2 + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// read value at target (shuffled) index
		uint iSrc = i * wordSize + offsetSrc;
		temp[i] = S[iSrc];
		temp[i + 1] = S[iSrc + 1];

		//S[iDst]	  = S[iSrc];
		//S[iDst + 1] = S[iSrc + 1];
	}
	uint offsetDst = idx * xStep + idy * yStep;
	for (uint i = 0; i < wordSize * 2; i += 2) {

		// write value to src (pre-shuffle) index
		uint iDst = i + offsetDst;
		S[iDst] = temp[i];
		S[iDst + 1] = temp[i + 1];
	}
}
// rotations and stuff
TEMPLATE_A __device__ void rotate(float* S)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;
	const float scaling = 2 * HIP_PI_F / fftSize;

	for (uint i = 0; i < wordSize; i++) {

		float a = (float)idx;	// floor(index / 8) tid is basically that, no need for more calculations
		float b = (float)i;		// mod(i, 8) i is already guarenteed to be between 0 and 8
		float phi = a * b;
		float ang = scaling * phi;
		float c = cosf(ang);
		float s = sinf(ang);

		uint index = idx * xStep + idy * yStep + i * 2;
		float real = S[index];
		float imag = S[index + 1];
		S[index]	 = c * real + s * imag;
		S[index + 1] = c * imag - s * real;
	}
}
TEMPLATE_A __device__ void execute_8point_fft(float* S)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	uint index = idx * xStep + idy * yStep;
	const float coef = sqrtf(2.0f) / 2.0f;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;

	// stage 1
	{
		// butterflies
		x0 =  S[index +  0] + S[index +  8]; // R
		x1 =  S[index +  1] + S[index +  9]; // I
		x8 =  S[index +  0] - S[index +  8]; // R
		x9 =  S[index +  1] - S[index +  9]; // I
				  			 
		x2 =  S[index +  2] + S[index + 10]; // R
		x3 =  S[index +  3] + S[index + 11]; // I
		x10 = S[index +  2] - S[index + 10]; // R
		x11 = S[index +  3] - S[index + 11]; // I
							 
		x4 =  S[index +  4] + S[index + 12]; // R
		x5 =  S[index +  5] + S[index + 13]; // I
		x12 = S[index +  5] - S[index + 13]; // R (swapped)
		x13 = S[index + 12] - S[index +  4]; // I (swapped)

		x6 =  S[index +  6] + S[index + 14]; // R
		x7 =  S[index +  7] + S[index + 15]; // I
		x14 = S[index +  6] - S[index + 14]; // R
		x15 = S[index +  7] - S[index + 15]; // I

		// rotations
		x10 = x10 * coef;
		x11 = x11 * coef;

		float temp = x10 + x11;
		x11 = x11 - x10;
		x10 = temp;


		x14 = x14 * coef;
		x15 = x15 * coef;

		temp = -x14 + x15;
		x15 = -x15 - x14;
		x14 = temp;
	}

	// stage 2
	{
		// butterflies
		float tempR;
		float tempI;
		tempR = x0 + x4; // R
		tempI = x1 + x5; // I
		x4 = x0 - x4; // R
		x5 = x1 - x5; // I
		x0 = tempR;
		x1 = tempI;

		tempR = x2 + x6; // R
		tempI = x3 + x7; // I
		float tempR2 = x3 - x7;
		x7 = x6 - x2; // I (swapped)
		x6 = tempR2; // R (swapped)
		x2 = tempR;
		x3 = tempI;

		tempR = x8 + x12; // R
		tempI = x9 + x13; // I
		x12 = x8 - x12; // R
		x13 = x9 - x13; // I
		x8 = tempR;
		x9 = tempI;

		tempR = x10 + x14; // R
		tempI = x11 + x15; // I
		tempR2 = x11 - x15;
		x15 = x14 - x10; // I (swapped)
		x14 = tempR2; // R (swapped)
		x10 = tempR;
		x11 = tempI;
	}

	// stage 3
	{
		// butterflies (with bit reversal)
		S[index +  0] =  x0 +  x2;
		S[index +  1] =  x1 +  x3;
		S[index +  2] =  x8 + x10;
		S[index +  3] =  x9 + x11;

		S[index +  4] =  x4 +  x6;
		S[index +  5] =  x5 +  x7;
		S[index +  6] = x12 + x14;
		S[index +  7] = x13 + x15;

		S[index +  8] =  x0 -  x2;
		S[index +  9] =  x1 -  x3;
		S[index + 10] =  x8 - x10;
		S[index + 11] =  x9 - x11;

		S[index + 12] =  x4 -  x6;
		S[index + 13] =  x5 -  x7;
		S[index + 14] = x12 - x14;
		S[index + 15] = x13 - x15;
	}
}
TEMPLATE_B __device__ void execute_8point_fft_shuffled(float* S)
{
	INDEXING_ALIASES;
	STEPPING_ALIASES;

	const float coef = sqrtf(2.0f) / 2.0f;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;

	// stage 1
	{
		// butterflies
		if constexpr (bShuffleInput) {
			uint offsetR = idx * 2 + idy * yStep;
			uint offsetI = offsetR + 1;
			x0 =  S[wordSize *  0 + offsetR] + S[wordSize *  8 + offsetR]; // R
			x1 =  S[wordSize *  0 + offsetI] + S[wordSize *  8 + offsetI]; // I
			x8 =  S[wordSize *  0 + offsetR] - S[wordSize *  8 + offsetR]; // R
			x9 =  S[wordSize *  0 + offsetI] - S[wordSize *  8 + offsetI]; // I

			x2 =  S[wordSize *  2 + offsetR] + S[wordSize * 10 + offsetR]; // R
			x3 =  S[wordSize *  2 + offsetI] + S[wordSize * 10 + offsetI]; // I
			x10 = S[wordSize *  2 + offsetR] - S[wordSize * 10 + offsetR]; // R
			x11 = S[wordSize *  2 + offsetI] - S[wordSize * 10 + offsetI]; // I

			x4 =  S[wordSize *  4 + offsetR] + S[wordSize * 12 + offsetR]; // R
			x5 =  S[wordSize *  4 + offsetI] + S[wordSize * 12 + offsetI]; // I
			x12 = S[wordSize *  4 + offsetI] - S[wordSize * 12 + offsetI]; // R (swapped)
			x13 = S[wordSize * 12 + offsetR] - S[wordSize *  4 + offsetR]; // I (swapped)

			x6 =  S[wordSize *  6 + offsetR] + S[wordSize * 14 + offsetR]; // R
			x7 =  S[wordSize *  6 + offsetI] + S[wordSize * 14 + offsetI]; // I
			x14 = S[wordSize *  6 + offsetR] - S[wordSize * 14 + offsetR]; // R
			x15 = S[wordSize *  6 + offsetI] - S[wordSize * 14 + offsetI]; // I
		}
		else {
			uint index = idx * xStep + idy * yStep;
			x0 =  S[index +  0] + S[index +  8]; // R
			x1 =  S[index +  1] + S[index +  9]; // I
			x8 =  S[index +  0] - S[index +  8]; // R
			x9 =  S[index +  1] - S[index +  9]; // I
				  			 
			x2 =  S[index +  2] + S[index + 10]; // R
			x3 =  S[index +  3] + S[index + 11]; // I
			x10 = S[index +  2] - S[index + 10]; // R
			x11 = S[index +  3] - S[index + 11]; // I
							 
			x4 =  S[index +  4] + S[index + 12]; // R
			x5 =  S[index +  5] + S[index + 13]; // I
			x12 = S[index +  5] - S[index + 13]; // R (swapped)
			x13 = S[index + 12] - S[index +  4]; // I (swapped)

			x6 =  S[index +  6] + S[index + 14]; // R
			x7 =  S[index +  7] + S[index + 15]; // I
			x14 = S[index +  6] - S[index + 14]; // R
			x15 = S[index +  7] - S[index + 15]; // I
		}

		// rotations
		x10 = x10 * coef;
		x11 = x11 * coef;

		float temp = x10 + x11;
		x11 = x11 - x10;
		x10 = temp;


		x14 = x14 * coef;
		x15 = x15 * coef;

		temp = -x14 + x15;
		x15 = -x15 - x14;
		x14 = temp;
	}

	// stage 2
	{
		// butterflies
		float tempR;
		float tempI;
		tempR = x0 + x4; // R
		tempI = x1 + x5; // I
		x4 = x0 - x4; // R
		x5 = x1 - x5; // I
		x0 = tempR;
		x1 = tempI;

		tempR = x2 + x6; // R
		tempI = x3 + x7; // I
		float tempR2 = x3 - x7;
		x7 = x6 - x2; // I (swapped)
		x6 = tempR2; // R (swapped)
		x2 = tempR;
		x3 = tempI;

		tempR = x8 + x12; // R
		tempI = x9 + x13; // I
		x12 = x8 - x12; // R
		x13 = x9 - x13; // I
		x8 = tempR;
		x9 = tempI;

		tempR = x10 + x14; // R
		tempI = x11 + x15; // I
		tempR2 = x11 - x15;
		x15 = x14 - x10; // I (swapped)
		x14 = tempR2; // R (swapped)
		x10 = tempR;
		x11 = tempI;
	}

	// stage 3
	{
		// butterflies (with bit reversal)
		if constexpr (bShuffleOutput) {

		}
		else {
			uint index = idx * xStep + idy * yStep;
			S[index +  0] =  x0 +  x2;
			S[index +  1] =  x1 +  x3;
			S[index +  2] =  x8 + x10;
			S[index +  3] =  x9 + x11;
					   
			S[index +  4] =  x4 +  x6;
			S[index +  5] =  x5 +  x7;
			S[index +  6] = x12 + x14;
			S[index +  7] = x13 + x15;
					   
			S[index +  8] =  x0 -  x2;
			S[index +  9] =  x1 -  x3;
			S[index + 10] =  x8 - x10;
			S[index + 11] =  x9 - x11;

			S[index + 12] =  x4 -  x6;
			S[index + 13] =  x5 -  x7;
			S[index + 14] = x12 - x14;
			S[index + 15] = x13 - x15;
		}
	}
}

// core kernel
__global__ void fft(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);


	// input shuffle
	// shuffleB(S);
	// executing first 8-point FFT
	// execute_8point_fft(S);
	execute_8point_fft_shuffled<8, 64, true, false>(S);

	// rotation + shuffle
	rotate(S);
	shuffleB(S);

	// executing second 8-point FFT
	execute_8point_fft(S);

	// output shuffle
	shuffleB(S);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}

int main()
{
	float* pIN;
	float IN[2 * INPUT_SIZE];
	float OUT[2 * INPUT_SIZE];

	for (int i = 0; i < INPUT_SIZE; i++)
	{
		// DEBUGGING for advanced indexing
		IN[2 * i] = i % 64;
		IN[2 * i + 1] = i % 64;
	}

	int memsize = 2 * INPUT_SIZE * sizeof(float);
	hipMalloc((void**)&pIN, memsize);
	hipMemcpy(pIN, IN, memsize, hipMemcpyHostToDevice);

	// Gets rid of false flags with IntelliSense
#ifdef __HIPCC__
	#define KERNEL_GRID(grid, block) <<< grid, block >>>
#else
	#define KERNEL_GRID(grid, block)
#endif
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(8, INPUT_SIZE / 64, 1);

	fft KERNEL_GRID(gridDim, blockDim)(pIN, pIN);
	hipMemcpy(OUT, pIN, memsize, hipMemcpyDeviceToHost);


	printf("The  outputs are: \n");
	for (int l = 0; l < INPUT_SIZE; l++) {
		printf("RE:A[%d]=%10.2f\t\t\t, IM: A[%d]=%10.2f\t\t\t \n ", 2 * l, OUT[2 * l], 2 * l + 1, OUT[2 * l + 1]);
	}

}
