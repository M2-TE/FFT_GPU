
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define INPUT_SIZE 64
#define WORD_SIZE 8
#define WORD_COMPLEX WORD_SIZE * 2

typedef unsigned int uint;

template <uint nBits>
__device__ uint reverse_bits(uint val)
{
	// NOTE: standard bit reversal either doesnt work on gpu
	// or is just really unperformant, so i used the CUDA intrinsic __brev() instead
	// however, it always operates on the full 32 bits of a value, so it needs to be manually adjusted
	// to work with only x bits (x = 7 bits in the 64-fft case)
	
	// shift bits to the major part
	// to only reverse a selected range of bits
	return __brev(val << (sizeof(uint) * 8 - nBits));
}
__device__ void mem_transfer(float* src, float* dst)
{
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const uint tid = threadIdx.x;

	// TODO: make this read data in coalescence (irrelevant with only one active thread)
	// note: "perfect" coalescence would require [threads = wordSize * 2 * threadsPerBlock]
	for (uint i = 0; i < 16; i++) {
		dst[tid * step + i] = src[tid * step + i];
	}
}
__device__ void debug_values(float* S)
{
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const uint tid = threadIdx.x;

	for (uint i = 0; i < 8; i++) {
		printf("[Thread %d Value %d]\tReal: %f\t\tImag: %f\n", tid, i, S[tid * step + i * 2], S[tid * step + i * 2 + 1]);
	}
}

/// deprecated atm
__device__ void execute_8point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const float coef = sqrtf(2.0f) / 2.0f;
	const unsigned int wordSize = 8;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 8]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 9]; // I
	float x8 = IN[tid * step + 0] - IN[tid * step + 8]; // R
	float x9 = IN[tid * step + 1] - IN[tid * step + 9]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 10]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 11]; // I
	float x10 = IN[tid * step + 2] - IN[tid * step + 10]; // R
	float x11 = IN[tid * step + 3] - IN[tid * step + 11]; // I

	float x4 = IN[tid * step + 4] + IN[tid * step + 12]; // R
	float x5 = IN[tid * step + 5] + IN[tid * step + 13]; // I
	float x12 = IN[tid * step + 5] - IN[tid * step + 13]; // R (swapped)
	float x13 = IN[tid * step + 12] - IN[tid * step + 4]; // I (swapped)

	float x6 = IN[tid * step + 6] + IN[tid * step + 14]; // R
	float x7 = IN[tid * step + 7] + IN[tid * step + 15]; // I
	float x14 = IN[tid * step + 6] - IN[tid * step + 14]; // R
	float x15 = IN[tid * step + 7] - IN[tid * step + 15]; // I

	// rotations
	x10 = x10 * coef;
	x11 = x11 * coef;

	float temp = x10 + x11;
	x11 = x11 - x10;
	x10 = temp;


	x14 = x14 * coef;
	x15 = x15 * coef;

	temp = -x14 + x15;
	x15 = -x15 - x14;
	x14 = temp;

	// stage 2
	// butterflies
	float tempR;
	float tempI;
	tempR = x0 + x4; // R
	tempI = x1 + x5; // I
	x4 = x0 - x4; // R
	x5 = x1 - x5; // I
	x0 = tempR;
	x1 = tempI;

	tempR = x2 + x6; // R
	tempI = x3 + x7; // I
	float tempR2 = x3 - x7;
	x7 = x6 - x2; // I (swapped)
	x6 = tempR2; // R (swapped)
	x2 = tempR;
	x3 = tempI;

	tempR = x8 + x12; // R
	tempI = x9 + x13; // I
	x12 = x8 - x12; // R
	x13 = x9 - x13; // I
	x8 = tempR;
	x9 = tempI;

	tempR = x10 + x14; // R
	tempI = x11 + x15; // I
	tempR2 = x11 - x15;
	x15 = x14 - x10; // I (swapped)
	x14 = tempR2; // R (swapped)
	x10 = tempR;
	x11 = tempI;

	// stage 3
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x8 + x10;
	IN[tid * step + 3] = x9 + x11;

	IN[tid * step + 4] = x4 + x6;
	IN[tid * step + 5] = x5 + x7;
	IN[tid * step + 6] = x12 + x14;
	IN[tid * step + 7] = x13 + x15;

	IN[tid * step + 8] = x0 - x2;
	IN[tid * step + 9] = x1 - x3;
	IN[tid * step + 10] = x8 - x10;
	IN[tid * step + 11] = x9 - x11;

	IN[tid * step + 12] = x4 - x6;
	IN[tid * step + 13] = x5 - x7;
	IN[tid * step + 14] = x12 - x14;
	IN[tid * step + 15] = x13 - x15;
}
__device__ void execute_4point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 4;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 4]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 5]; // I
	float x4 = IN[tid * step + 0] - IN[tid * step + 4]; // R
	float x5 = IN[tid * step + 1] - IN[tid * step + 5]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 6]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 7]; // I
	float x6 = IN[tid * step + 3] - IN[tid * step + 7]; // R (swapped)
	float x7 = IN[tid * step + 6] - IN[tid * step + 2]; // I (swapped)

	// stage 2
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x4 + x6;
	IN[tid * step + 3] = x5 + x7;

	IN[tid * step + 4] = x0 - x2;
	IN[tid * step + 5] = x1 - x3;
	IN[tid * step + 6] = x4 - x6;
	IN[tid * step + 7] = x5 - x7;
}
__device__ void execute_2point_fft_deprecated(float* IN)
{
	//__shared__ float S[4];
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 2;
	const unsigned int step = wordSize * 2;

	// only really need to store 0 and 1 in registers
	// but compiler puts em all in registers anyways 
	// and this is just more readable:
	float x0 = IN[tid * step + 0];
	float x1 = IN[tid * step + 1];
	float x2 = IN[tid * step + 2];
	float x3 = IN[tid * step + 3];

	// stage 1
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x0 - x2;
	IN[tid * step + 3] = x1 - x3;
}
///

__device__ void execute_8point_fft(float* S)
{
	const uint tid = threadIdx.x;
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const float coef = sqrtf(2.0f) / 2.0f;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;

	// stage 1
	{
		// butterflies
		x0  = S[tid * step +  0] + S[tid * step +  8]; // R
		x1  = S[tid * step +  1] + S[tid * step +  9]; // I
		x8  = S[tid * step +  0] - S[tid * step +  8]; // R
		x9  = S[tid * step +  1] - S[tid * step +  9]; // I
		
		x2  = S[tid * step +  2] + S[tid * step + 10]; // R
		x3  = S[tid * step +  3] + S[tid * step + 11]; // I
		x10 = S[tid * step +  2] - S[tid * step + 10]; // R
		x11 = S[tid * step +  3] - S[tid * step + 11]; // I

		x4  = S[tid * step +  4] + S[tid * step + 12]; // R
		x5  = S[tid * step +  5] + S[tid * step + 13]; // I
		x12 = S[tid * step +  5] - S[tid * step + 13]; // R (swapped)
		x13 = S[tid * step + 12] - S[tid * step +  4]; // I (swapped)

		x6  = S[tid * step +  6] + S[tid * step + 14]; // R
		x7  = S[tid * step +  7] + S[tid * step + 15]; // I
		x14 = S[tid * step +  6] - S[tid * step + 14]; // R
		x15 = S[tid * step +  7] - S[tid * step + 15]; // I

		// rotations
		x10 = x10 * coef;
		x11 = x11 * coef;

		float temp = x10 + x11;
		x11 = x11 - x10;
		x10 = temp;


		x14 = x14 * coef;
		x15 = x15 * coef;

		temp = -x14 + x15;
		x15 = -x15 - x14;
		x14 = temp;
	}

	// stage 2
	{
		// butterflies
		float tempR;
		float tempI;
		tempR = x0 + x4; // R
		tempI = x1 + x5; // I
		x4 = x0 - x4; // R
		x5 = x1 - x5; // I
		x0 = tempR;
		x1 = tempI;

		tempR = x2 + x6; // R
		tempI = x3 + x7; // I
		float tempR2 = x3 - x7;
		x7 = x6 - x2; // I (swapped)
		x6 = tempR2; // R (swapped)
		x2 = tempR;
		x3 = tempI;

		tempR = x8 + x12; // R
		tempI = x9 + x13; // I
		x12 = x8 - x12; // R
		x13 = x9 - x13; // I
		x8 = tempR;
		x9 = tempI;

		tempR = x10 + x14; // R
		tempI = x11 + x15; // I
		tempR2 = x11 - x15;
		x15 = x14 - x10; // I (swapped)
		x14 = tempR2; // R (swapped)
		x10 = tempR;
		x11 = tempI;
	}

	// stage 3
	{
		// butterflies (with bit reversal)
		S[tid * step + 0] = x0 + x2;
		S[tid * step + 1] = x1 + x3;
		S[tid * step + 2] = x8 + x10;
		S[tid * step + 3] = x9 + x11;

		S[tid * step + 4] = x4 + x6;
		S[tid * step + 5] = x5 + x7;
		S[tid * step + 6] = x12 + x14;
		S[tid * step + 7] = x13 + x15;

		S[tid * step + 8] = x0 - x2;
		S[tid * step + 9] = x1 - x3;
		S[tid * step + 10] = x8 - x10;
		S[tid * step + 11] = x9 - x11;

		S[tid * step + 12] = x4 - x6;
		S[tid * step + 13] = x5 - x7;
		S[tid * step + 14] = x12 - x14;
		S[tid * step + 15] = x13 - x15;
	}
}
__device__ void shuffle(float* S)
{
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const uint tid = threadIdx.x;

	// stuff needed to read the values to swap with
	const uint tidLocal = tid % WORD_SIZE; // tid within 64-point fft
	const uint offset = tidLocal > 3 ? tid - 4 : tid + 4;

	uint start = tid * step;
	uint other = offset * step;
	for (uint i = 0; i < WORD_COMPLEX; i++) {

		// read other value into register first
		float val = S[other + i];
		// then write to own thread-local value, effectively swapping
		S[start + i] = val;
	}
}
__device__ void rotate(float* S, uint index)
{
	float pi = 3.14;
	float scaling = 2 * pi / 64;

	float a = (float)index / 8.0f;
	float b = (float)(index % 8);
	float phi = floorf(a * b);
	float ang = scaling * phi;
	float c = cosf(ang);
	float s = sinf(ang);

	// TODO
}
__global__ void fft(float* IN, float* OUT)
{
	__shared__ float S[INPUT_SIZE * 2];

	// transfer from global to shared memory
	mem_transfer(IN, S);


	// input shuffle
	debug_values(S);
	shuffle(S);
	debug_values(S);

	return;
	// executing first 8-point FFT
	execute_8point_fft(S);

	// rotation + shuffle or shuffle + rotation
	rotate(S, 0); // index = 0, TODO
	shuffle(S);

	// executing second 8-point FFT
	execute_8point_fft(S);

	// output shuffle
	shuffle(S);

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}

int main()
{
	//uint val = 0b0100'0001;
	//printf("%d\n", reverse_bits<7>(val));
	//return;

	static constexpr size_t N = INPUT_SIZE;

	float A[2 * N];
	float* Ad;

	int memsize = 2 * N * sizeof(float);


	for (int i = 0; i < N; i++)
	{
		A[2 * i] = i;
		A[2 * i + 1] = i;
	}


	hipMalloc((void**)&Ad, memsize);

	hipMemcpy(Ad, A, memsize, hipMemcpyHostToDevice);


	// Gets rid of false flags with IntelliSense
#ifdef __HIPCC__
	#define KERNEL_GRID(grid, block) <<< grid, block >>>
#else
	#define KERNEL_GRID(grid, block)
#endif
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(INPUT_SIZE / WORD_SIZE, 1, 1);
	fft KERNEL_GRID(gridDim, blockDim)(Ad, Ad);
	hipMemcpy(A, Ad, memsize, hipMemcpyDeviceToHost);


	printf("The  outputs are: \n");
	for (int l = 0; l < N; l++) {
		//printf("RE:A[%d]=%10.2f\t\t\t, IM: A[%d]=%10.2f\t\t\t \n ", 2 * l, A[2 * l], 2 * l + 1, A[2 * l + 1]);
	}

}
