
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

typedef unsigned int uint;

template <uint nBits>
__device__ uint reverse_bits(uint val)
{
	// NOTE: standard bit reversal either doesnt work on gpu
	// or is just really unperformant, so i used the CUDA intrinsic __brev() instead
	// however, it always operates on the full 32 bits of a value, so it needs to be manually adjusted
	// to work with only x bits (x = 7 bits in the 64-fft case)
	
	// shift bits to the major part
	// to only reverse a selected range of bits
	return __brev(val << (sizeof(uint) * 8 - nBits));
}
__device__ void mem_transfer(float* src, float* dst)
{
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const uint tid = threadIdx.x;

	// TODO: make this read data in coalescence (irrelevant with only one active thread)
	// note: "perfect" coalescence would require [threads = wordSize * 2 * threadsPerBlock]
	for (uint i = 0; i < 16; i++) {
		dst[tid * step + i] = src[tid * step + i];
	}
}

/// deprecated atm
__device__ void execute_8point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const float coef = sqrtf(2.0f) / 2.0f;
	const unsigned int wordSize = 8;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 8]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 9]; // I
	float x8 = IN[tid * step + 0] - IN[tid * step + 8]; // R
	float x9 = IN[tid * step + 1] - IN[tid * step + 9]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 10]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 11]; // I
	float x10 = IN[tid * step + 2] - IN[tid * step + 10]; // R
	float x11 = IN[tid * step + 3] - IN[tid * step + 11]; // I

	float x4 = IN[tid * step + 4] + IN[tid * step + 12]; // R
	float x5 = IN[tid * step + 5] + IN[tid * step + 13]; // I
	float x12 = IN[tid * step + 5] - IN[tid * step + 13]; // R (swapped)
	float x13 = IN[tid * step + 12] - IN[tid * step + 4]; // I (swapped)

	float x6 = IN[tid * step + 6] + IN[tid * step + 14]; // R
	float x7 = IN[tid * step + 7] + IN[tid * step + 15]; // I
	float x14 = IN[tid * step + 6] - IN[tid * step + 14]; // R
	float x15 = IN[tid * step + 7] - IN[tid * step + 15]; // I

	// rotations
	x10 = x10 * coef;
	x11 = x11 * coef;

	float temp = x10 + x11;
	x11 = x11 - x10;
	x10 = temp;


	x14 = x14 * coef;
	x15 = x15 * coef;

	temp = -x14 + x15;
	x15 = -x15 - x14;
	x14 = temp;

	// stage 2
	// butterflies
	float tempR;
	float tempI;
	tempR = x0 + x4; // R
	tempI = x1 + x5; // I
	x4 = x0 - x4; // R
	x5 = x1 - x5; // I
	x0 = tempR;
	x1 = tempI;

	tempR = x2 + x6; // R
	tempI = x3 + x7; // I
	float tempR2 = x3 - x7;
	x7 = x6 - x2; // I (swapped)
	x6 = tempR2; // R (swapped)
	x2 = tempR;
	x3 = tempI;

	tempR = x8 + x12; // R
	tempI = x9 + x13; // I
	x12 = x8 - x12; // R
	x13 = x9 - x13; // I
	x8 = tempR;
	x9 = tempI;

	tempR = x10 + x14; // R
	tempI = x11 + x15; // I
	tempR2 = x11 - x15;
	x15 = x14 - x10; // I (swapped)
	x14 = tempR2; // R (swapped)
	x10 = tempR;
	x11 = tempI;

	// stage 3
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x8 + x10;
	IN[tid * step + 3] = x9 + x11;

	IN[tid * step + 4] = x4 + x6;
	IN[tid * step + 5] = x5 + x7;
	IN[tid * step + 6] = x12 + x14;
	IN[tid * step + 7] = x13 + x15;

	IN[tid * step + 8] = x0 - x2;
	IN[tid * step + 9] = x1 - x3;
	IN[tid * step + 10] = x8 - x10;
	IN[tid * step + 11] = x9 - x11;

	IN[tid * step + 12] = x4 - x6;
	IN[tid * step + 13] = x5 - x7;
	IN[tid * step + 14] = x12 - x14;
	IN[tid * step + 15] = x13 - x15;
}
__device__ void execute_4point_fft_deprecated(float* IN)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 4;
	const unsigned int step = wordSize * 2;

	// stage 1
	// butterflies
	float x0 = IN[tid * step + 0] + IN[tid * step + 4]; // R
	float x1 = IN[tid * step + 1] + IN[tid * step + 5]; // I
	float x4 = IN[tid * step + 0] - IN[tid * step + 4]; // R
	float x5 = IN[tid * step + 1] - IN[tid * step + 5]; // I

	float x2 = IN[tid * step + 2] + IN[tid * step + 6]; // R
	float x3 = IN[tid * step + 3] + IN[tid * step + 7]; // I
	float x6 = IN[tid * step + 3] - IN[tid * step + 7]; // R (swapped)
	float x7 = IN[tid * step + 6] - IN[tid * step + 2]; // I (swapped)

	// stage 2
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x4 + x6;
	IN[tid * step + 3] = x5 + x7;

	IN[tid * step + 4] = x0 - x2;
	IN[tid * step + 5] = x1 - x3;
	IN[tid * step + 6] = x4 - x6;
	IN[tid * step + 7] = x5 - x7;
}
__device__ void execute_2point_fft_deprecated(float* IN)
{
	//__shared__ float S[4];
	const unsigned int tid = threadIdx.x;
	const unsigned int wordSize = 2;
	const unsigned int step = wordSize * 2;

	// only really need to store 0 and 1 in registers
	// but compiler puts em all in registers anyways 
	// and this is just more readable:
	float x0 = IN[tid * step + 0];
	float x1 = IN[tid * step + 1];
	float x2 = IN[tid * step + 2];
	float x3 = IN[tid * step + 3];

	// stage 1
	// butterflies
	IN[tid * step + 0] = x0 + x2;
	IN[tid * step + 1] = x1 + x3;
	IN[tid * step + 2] = x0 - x2;
	IN[tid * step + 3] = x1 - x3;
}
///

__device__ void execute_8point_fft(float* S)
{
	const uint tid = threadIdx.x;
	const uint wordSize = 8;
	const uint step = wordSize * 2;
	const float coef = sqrtf(2.0f) / 2.0f;

	// registers for the main data inbetween stages
	float x0, x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15;

	// stage 1
	{
		// butterflies
		x0  = S[tid * step +  0] + S[tid * step +  8]; // R
		x1  = S[tid * step +  1] + S[tid * step +  9]; // I
		x8  = S[tid * step +  0] - S[tid * step +  8]; // R
		x9  = S[tid * step +  1] - S[tid * step +  9]; // I
		
		x2  = S[tid * step +  2] + S[tid * step + 10]; // R
		x3  = S[tid * step +  3] + S[tid * step + 11]; // I
		x10 = S[tid * step +  2] - S[tid * step + 10]; // R
		x11 = S[tid * step +  3] - S[tid * step + 11]; // I

		x4  = S[tid * step +  4] + S[tid * step + 12]; // R
		x5  = S[tid * step +  5] + S[tid * step + 13]; // I
		x12 = S[tid * step +  5] - S[tid * step + 13]; // R (swapped)
		x13 = S[tid * step + 12] - S[tid * step +  4]; // I (swapped)

		x6  = S[tid * step +  6] + S[tid * step + 14]; // R
		x7  = S[tid * step +  7] + S[tid * step + 15]; // I
		x14 = S[tid * step +  6] - S[tid * step + 14]; // R
		x15 = S[tid * step +  7] - S[tid * step + 15]; // I

		// rotations
		x10 = x10 * coef;
		x11 = x11 * coef;

		float temp = x10 + x11;
		x11 = x11 - x10;
		x10 = temp;


		x14 = x14 * coef;
		x15 = x15 * coef;

		temp = -x14 + x15;
		x15 = -x15 - x14;
		x14 = temp;
	}

	// stage 2
	{
		// butterflies
		float tempR;
		float tempI;
		tempR = x0 + x4; // R
		tempI = x1 + x5; // I
		x4 = x0 - x4; // R
		x5 = x1 - x5; // I
		x0 = tempR;
		x1 = tempI;

		tempR = x2 + x6; // R
		tempI = x3 + x7; // I
		float tempR2 = x3 - x7;
		x7 = x6 - x2; // I (swapped)
		x6 = tempR2; // R (swapped)
		x2 = tempR;
		x3 = tempI;

		tempR = x8 + x12; // R
		tempI = x9 + x13; // I
		x12 = x8 - x12; // R
		x13 = x9 - x13; // I
		x8 = tempR;
		x9 = tempI;

		tempR = x10 + x14; // R
		tempI = x11 + x15; // I
		tempR2 = x11 - x15;
		x15 = x14 - x10; // I (swapped)
		x14 = tempR2; // R (swapped)
		x10 = tempR;
		x11 = tempI;
	}

	// stage 3
	{
		// butterflies (with bit reversal)
		S[tid * step + 0] = x0 + x2;
		S[tid * step + 1] = x1 + x3;
		S[tid * step + 2] = x8 + x10;
		S[tid * step + 3] = x9 + x11;

		S[tid * step + 4] = x4 + x6;
		S[tid * step + 5] = x5 + x7;
		S[tid * step + 6] = x12 + x14;
		S[tid * step + 7] = x13 + x15;

		S[tid * step + 8] = x0 - x2;
		S[tid * step + 9] = x1 - x3;
		S[tid * step + 10] = x8 - x10;
		S[tid * step + 11] = x9 - x11;

		S[tid * step + 12] = x4 - x6;
		S[tid * step + 13] = x5 - x7;
		S[tid * step + 14] = x12 - x14;
		S[tid * step + 15] = x13 - x15;
	}
}
__device__ void shuffle()
{

}
__device__ void rotate()
{

}
__global__ void fft(float* IN, float* OUT)
{
	__shared__ float S[64];

	// transfer from global to shared memory
	mem_transfer(IN, S);

	// input shuffle
	shuffle();
	
	// executing first 8-point FFT
	execute_8point_fft(S);

	// rotation + shuffle or shuffle + rotation
	rotate();
	shuffle();

	// executing second 8-point FFT
	execute_8point_fft(S);

	// output shuffle
	shuffle();

	// transfer from shared to global memory
	mem_transfer(S, OUT);
}

#define INPUT_SIZE 64
#define WORD_SIZE 8
int main()
{
	//uint val = 0b0100'0001;
	//printf("%d\n", reverse_bits<7>(val));
	//return;

	static constexpr size_t N = INPUT_SIZE;

	float A[2 * N];
	float* Ad;

	int memsize = 2 * N * sizeof(float);


	for (int i = 0; i < N; i++)
	{
		A[2 * i] = i;
		A[2 * i + 1] = i;
	}


	hipMalloc((void**)&Ad, memsize);

	hipMemcpy(Ad, A, memsize, hipMemcpyHostToDevice);


	// Gets rid of false flags with IntelliSense
#ifdef __HIPCC__
	#define KERNEL_GRID(grid, block) <<< grid, block >>>
#else
	#define KERNEL_GRID(grid, block)
#endif
	dim3 gridDim(1, 1, 1);
	dim3 blockDim(INPUT_SIZE / WORD_SIZE, 1, 1);
	fft KERNEL_GRID(gridDim, blockDim)(Ad, Ad);
	hipMemcpy(A, Ad, memsize, hipMemcpyDeviceToHost);


	printf("The  outputs are: \n");
	for (int l = 0; l < N; l++) {
		printf("RE:A[%d]=%10.2f\t\t\t, IM: A[%d]=%10.2f\t\t\t \n ", 2 * l, A[2 * l], 2 * l + 1, A[2 * l + 1]);
	}

}
