#include "hip/hip_runtime.h"
#define N 512 // N complex numbers
#define M 2*N // M 32-bit floats (real, imaginary)
#define D N/8 // D number of threads (one block), each thread processing an 8-point fft   
__global__  void fft(float* A, float* ROT)
{
	__shared__ float SA[M], SB[M], SROT[N];
	short i = threadIdx.x;
	short n = logf(N) / logf(8);

	SROT[i] = ROT[i];
	SROT[i + blockDim.x] = ROT[i + blockDim.x];
	SROT[i + 2 * blockDim.x] = ROT[i + 2 * blockDim.x];
	SROT[i + 3 * blockDim.x] = ROT[i + 3 * blockDim.x];
	SROT[i + 4 * blockDim.x] = ROT[i + 4 * blockDim.x];
	SROT[i + 5 * blockDim.x] = ROT[i + 5 * blockDim.x];
	SROT[i + 6 * blockDim.x] = ROT[i + 6 * blockDim.x];
	SROT[i + 7 * blockDim.x] = ROT[i + 7 * blockDim.x];

	SA[i] = A[i];
	SA[i + blockDim.x] = A[i + blockDim.x];
	SA[i + 2 * blockDim.x] = A[i + 2 * blockDim.x];
	SA[i + 3 * blockDim.x] = A[i + 3 * blockDim.x];
	SA[i + 4 * blockDim.x] = A[i + 4 * blockDim.x];
	SA[i + 5 * blockDim.x] = A[i + 5 * blockDim.x];
	SA[i + 6 * blockDim.x] = A[i + 6 * blockDim.x];
	SA[i + 7 * blockDim.x] = A[i + 7 * blockDim.x];
	SA[i + 8 * blockDim.x] = A[i + 8 * blockDim.x];
	SA[i + 9 * blockDim.x] = A[i + 9 * blockDim.x];
	SA[i + 10 * blockDim.x] = A[i + 10 * blockDim.x];
	SA[i + 11 * blockDim.x] = A[i + 11 * blockDim.x];
	SA[i + 12 * blockDim.x] = A[i + 12 * blockDim.x];
	SA[i + 13 * blockDim.x] = A[i + 13 * blockDim.x];
	SA[i + 14 * blockDim.x] = A[i + 14 * blockDim.x];
	SA[i + 15 * blockDim.x] = A[i + 15 * blockDim.x];

	/*
	0R	0
	0I	1
	1R	2
	1I	3
		...
	7I	15

	//1st stage:
	temp = SA[0] + SA[8];
	SA[8] = SA[0] - SA[8];
	SA[0] = temp;

	...
	and so on

	temp = SA[4] + SA[12];
	// simplified here on phi=2
	*/

	__syncthreads();

	short ind0, ind1, ind2, ind3, ind4, ind5, ind6, ind7;
	short r0, r1, r2, r3, r4, r5, r6, r7, r8, r9, r10, r11;

	for (short s = 1; s <= n; s++)
	{
		short p = M / (1 << (3 * s));
		ind0 = 2 * (i + (i / (1 << 3 * (n - s))) * (1 << 3 * (n - s)) * 7);
		ind1 = ind0 + p;
		ind2 = ind1 + p;//ind0+2p
		ind3 = ind2 + p;//ind0+3p
		ind4 = ind3 + p;//ind0+4p
		ind5 = ind4 + p;//ind0+5p
		ind6 = ind5 + p;//ind0+6p
		ind7 = ind6 + p;//ind0+7p

		r0 = (i % (1 << 3 * (n - s))) * (1 << 3 * (s - 1));
		r1 = r0 + (N / 8);
		r2 = r1 + (N / 8);//r0+2(N/8)
		r3 = r2 + (N / 8);//r0+3(N/8)
		r4 = 2 * r0;
		r5 = r4 + (N / 4);
		r6 = r4;
		r7 = r5;
		r8 = 2 * r4;
		r9 = r8;
		r10 = r8;
		r11 = r8;
		/*SA[i]    = ind0;
		SA[i+8]  = ind1;
		SA[i+16] = ind2;
		SA[i+24] = ind3;
		SA[i+32] = ind4;
		SA[i+40] = ind5;
		SA[i+48] = ind6;
		SA[i+54] = ind7;*/

		/*SA[i]    = r0;
		SA[i+8]  = r1;
		SA[i+16] = r2;
		SA[i+24] = r3;
		SA[i+32] = r4;
		SA[i+40] = r5;
		SA[i+48] = r6;
		SA[i+56] = r7;
		SA[i+64] = r8;
		SA[i+72] = r9;
		SA[i+80] = r10;
		SA[i+88] = r11;*/

		//1st stage:
		SB[ind0] = SA[ind0] + SA[ind4];
		SB[ind0 + 1] = SA[ind0 + 1] + SA[ind4 + 1];
		SB[ind4] = SA[ind0] - SA[ind4];
		SB[ind4 + 1] = SA[ind0 + 1] - SA[ind4 + 1];

		SB[ind1] = SA[ind1] + SA[ind5];
		SB[ind1 + 1] = SA[ind1 + 1] + SA[ind5 + 1];
		SB[ind5] = SA[ind1] - SA[ind5];
		SB[ind5 + 1] = SA[ind1 + 1] - SA[ind5 + 1];

		SB[ind2] = SA[ind2] + SA[ind6];
		SB[ind2 + 1] = SA[ind2 + 1] + SA[ind6 + 1];
		SB[ind6] = SA[ind2] - SA[ind6];
		SB[ind6 + 1] = SA[ind2 + 1] - SA[ind6 + 1];

		SB[ind3] = SA[ind3] + SA[ind7];
		SB[ind3 + 1] = SA[ind3 + 1] + SA[ind7 + 1];
		SB[ind7] = SA[ind3] - SA[ind7];
		SB[ind7 + 1] = SA[ind3 + 1] - SA[ind7 + 1];

		SA[ind0] = SB[ind0];
		SA[ind0 + 1] = SB[ind0 + 1];
		SA[ind4] = SB[ind4] * SROT[2 * r0] + SB[ind4 + 1] * SROT[2 * r0 + 1];
		SA[ind4 + 1] = -SB[ind4] * SROT[2 * r0 + 1] + SB[ind4 + 1] * SROT[2 * r0];

		SA[ind1] = SB[ind1];
		SA[ind1 + 1] = SB[ind1 + 1];
		SA[ind5] = SB[ind5] * SROT[2 * r1] + SB[ind5 + 1] * SROT[2 * r1 + 1];
		SA[ind5 + 1] = -SB[ind5] * SROT[2 * r1 + 1] + SB[ind5 + 1] * SROT[2 * r1];

		SA[ind2] = SB[ind2];
		SA[ind2 + 1] = SB[ind2 + 1];
		SA[ind6] = SB[ind6] * SROT[2 * r2] + SB[ind6 + 1] * SROT[2 * r2 + 1];
		SA[ind6 + 1] = -SB[ind6] * SROT[2 * r2 + 1] + SB[ind6 + 1] * SROT[2 * r2];

		SA[ind3] = SB[ind3];
		SA[ind3 + 1] = SB[ind3 + 1];
		SA[ind7] = SB[ind7] * SROT[2 * r3] + SB[ind7 + 1] * SROT[2 * r3 + 1];
		SA[ind7 + 1] = -SB[ind7] * SROT[2 * r3 + 1] + SB[ind7 + 1] * SROT[2 * r3];
		__syncthreads();

		//2nd stage:
		SB[ind0] = SA[ind0] + SA[ind2];
		SB[ind0 + 1] = SA[ind0 + 1] + SA[ind2 + 1];
		SB[ind2] = SA[ind0] - SA[ind2];
		SB[ind2 + 1] = SA[ind0 + 1] - SA[ind2 + 1];

		SB[ind1] = SA[ind1] + SA[ind3];
		SB[ind1 + 1] = SA[ind1 + 1] + SA[ind3 + 1];
		SB[ind3] = SA[ind1] - SA[ind3];
		SB[ind3 + 1] = SA[ind1 + 1] - SA[ind3 + 1];

		SB[ind4] = SA[ind4] + SA[ind6];
		SB[ind4 + 1] = SA[ind4 + 1] + SA[ind6 + 1];
		SB[ind6] = SA[ind4] - SA[ind6];
		SB[ind6 + 1] = SA[ind4 + 1] - SA[ind6 + 1];

		SB[ind5] = SA[ind5] + SA[ind7];
		SB[ind5 + 1] = SA[ind5 + 1] + SA[ind7 + 1];
		SB[ind7] = SA[ind5] - SA[ind7];
		SB[ind7 + 1] = SA[ind5 + 1] - SA[ind7 + 1];

		SA[ind0] = SB[ind0];
		SA[ind0 + 1] = SB[ind0 + 1];
		SA[ind2] = SB[ind2] * SROT[2 * r4] + SB[ind2 + 1] * SROT[2 * r4 + 1];
		SA[ind2 + 1] = -SB[ind2] * SROT[2 * r4 + 1] + SB[ind2 + 1] * SROT[2 * r4];

		SA[ind1] = SB[ind1];
		SA[ind1 + 1] = SB[ind1 + 1];
		SA[ind3] = SB[ind3] * SROT[2 * r5] + SB[ind3 + 1] * SROT[2 * r5 + 1];
		SA[ind3 + 1] = -SB[ind3] * SROT[2 * r5 + 1] + SB[ind3 + 1] * SROT[2 * r5];

		SA[ind4] = SB[ind4];
		SA[ind4 + 1] = SB[ind4 + 1];
		SA[ind6] = SB[ind6] * SROT[2 * r6] + SB[ind6 + 1] * SROT[2 * r6 + 1];
		SA[ind6 + 1] = -SB[ind6] * SROT[2 * r6 + 1] + SB[ind6 + 1] * SROT[2 * r6];

		SA[ind5] = SB[ind5];
		SA[ind5 + 1] = SB[ind5 + 1];
		SA[ind7] = SB[ind7] * SROT[2 * r7] + SB[ind7 + 1] * SROT[2 * r7 + 1];
		SA[ind7 + 1] = -SB[ind7] * SROT[2 * r7 + 1] + SB[ind7 + 1] * SROT[2 * r7];
		__syncthreads();
		//3rd stage:
		SB[ind0] = SA[ind0] + SA[ind1];
		SB[ind0 + 1] = SA[ind0 + 1] + SA[ind1 + 1];
		SB[ind1] = SA[ind0] - SA[ind1];
		SB[ind1 + 1] = SA[ind0 + 1] - SA[ind1 + 1];

		SB[ind2] = SA[ind2] + SA[ind3];
		SB[ind2 + 1] = SA[ind2 + 1] + SA[ind3 + 1];
		SB[ind3] = SA[ind2] - SA[ind3];
		SB[ind3 + 1] = SA[ind2 + 1] - SA[ind3 + 1];

		SB[ind4] = SA[ind4] + SA[ind5];
		SB[ind4 + 1] = SA[ind4 + 1] + SA[ind5 + 1];
		SB[ind5] = SA[ind4] - SA[ind5];
		SB[ind5 + 1] = SA[ind4 + 1] - SA[ind5 + 1];

		SB[ind6] = SA[ind6] + SA[ind7];
		SB[ind6 + 1] = SA[ind6 + 1] + SA[ind7 + 1];
		SB[ind7] = SA[ind6] - SA[ind7];
		SB[ind7 + 1] = SA[ind6 + 1] - SA[ind7 + 1];

		SA[ind0] = SB[ind0];
		SA[ind0 + 1] = SB[ind0 + 1];
		SA[ind1] = SB[ind1] * SROT[2 * r8] + SB[ind1 + 1] * SROT[2 * r8 + 1];
		SA[ind1 + 1] = -SB[ind1] * SROT[2 * r8 + 1] + SB[ind1 + 1] * SROT[2 * r8];

		SA[ind2] = SB[ind2];
		SA[ind2 + 1] = SB[ind2 + 1];
		SA[ind3] = SB[ind3] * SROT[2 * r9] + SB[ind3 + 1] * SROT[2 * r9 + 1];
		SA[ind3 + 1] = -SB[ind3] * SROT[2 * r9 + 1] + SB[ind3 + 1] * SROT[2 * r9];

		SA[ind4] = SB[ind4];
		SA[ind4 + 1] = SB[ind4 + 1];
		SA[ind5] = SB[ind5] * SROT[2 * r10] + SB[ind5 + 1] * SROT[2 * r10 + 1];
		SA[ind5 + 1] = -SB[ind5] * SROT[2 * r10 + 1] + SB[ind5 + 1] * SROT[2 * r10];

		SA[ind6] = SB[ind6];
		SA[ind6 + 1] = SB[ind6 + 1];
		SA[ind7] = SB[ind7] * SROT[2 * r11] + SB[ind7 + 1] * SROT[2 * r11 + 1];
		SA[ind7 + 1] = -SB[ind7] * SROT[2 * r11 + 1] + SB[ind7 + 1] * SROT[2 * r11];
		__syncthreads();

	}

	A[i] = SA[i];
	A[i + blockDim.x] = SA[i + blockDim.x];
	A[i + 2 * blockDim.x] = SA[i + 2 * blockDim.x];
	A[i + 3 * blockDim.x] = SA[i + 3 * blockDim.x];
	A[i + 4 * blockDim.x] = SA[i + 4 * blockDim.x];
	A[i + 5 * blockDim.x] = SA[i + 5 * blockDim.x];
	A[i + 6 * blockDim.x] = SA[i + 6 * blockDim.x];
	A[i + 7 * blockDim.x] = SA[i + 7 * blockDim.x];
	A[i + 8 * blockDim.x] = SA[i + 8 * blockDim.x];
	A[i + 9 * blockDim.x] = SA[i + 9 * blockDim.x];
	A[i + 10 * blockDim.x] = SA[i + 10 * blockDim.x];
	A[i + 11 * blockDim.x] = SA[i + 11 * blockDim.x];
	A[i + 12 * blockDim.x] = SA[i + 12 * blockDim.x];
	A[i + 13 * blockDim.x] = SA[i + 13 * blockDim.x];
	A[i + 14 * blockDim.x] = SA[i + 14 * blockDim.x];
	A[i + 15 * blockDim.x] = SA[i + 15 * blockDim.x];
	__syncthreads();
}


#include  <stdio.h> 
#include  <math.h>
int  main()
{

	float A[2 * N];
	float* Ad;
	float ROT[N];
	float* ROTd;

	int memsize = 2 * N * sizeof(float);
	int rotsize = N * sizeof(float);


	for (int i = 0; i < N; i++)
	{
		A[2 * i] = i;
		A[2 * i + 1] = i;
	}
	for (int j = 0; j < (N / 2); j++)
	{

		ROT[2 * j] = cosf((j * (6.2857)) / N);
		ROT[2 * j + 1] = sinf((j * (6.2857)) / N);
	}

	hipMalloc((void**)&Ad, memsize);
	hipMalloc((void**)&ROTd, rotsize);

	hipMemcpy(Ad, A, memsize, hipMemcpyHostToDevice);
	hipMemcpy(ROTd, ROT, rotsize, hipMemcpyHostToDevice);

	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter); 
	dim3 gridDim(1, 1);
	dim3 blockDim(D, 1);
	fft << <gridDim, blockDim >> > (Ad, ROTd);
	hipMemcpy(A, Ad, memsize, hipMemcpyDeviceToHost);
	hipMemcpy(ROT, ROTd, rotsize, hipMemcpyDeviceToHost);


	int i = 63;
	int n = logf(N) / logf(8);
	for (int s = 1; s <= n; s++) {

		int b = 2 * (i + (i / (1 << 3 * (n - s))) * (1 << 3 * (n - s)) * 7);
		int p = M / (1 << (3 * s));

		printf("%d\n", b + 7 * p);
	}

	//printf("The  outputs are: \n");
	//for (int l = 0; l < N; l++)
	//	printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ", 2 * l, A[2 * l], 2 * l + 1, A[2 * l + 1]);

}
