	#define N 1024
	#define M 2*N
	#define D N/2
	__global__  void fft(float* A , float* ROT) 
       { 
	__shared__ float SA[N],SB[N],SROT[M];
	short j   = threadIdx.x;
	short n = logf(N)/logf(2);
	SA[j] = A[j+blockIdx.x*(N>>1)];
	SA[j+blockDim.x] = A[j+blockIdx.x*(N>>1)+N];
	SROT[j] = ROT[j];
	SROT[j+blockDim.x] = ROT[j+blockDim.x];
	SROT[j+2*blockDim.x] = ROT[j+2*blockDim.x];
	SROT[j+3*blockDim.x] = ROT[j+3*blockDim.x];
	__syncthreads();
	short g = j+blockIdx.x*blockDim.x;
	 short i = j>>1;
    	short k = j%2;
   
    	short ind0 = i<<1;
    	short ind1 = (i<<1) + (N>>1);
    	short ind2 = j<<1;
    	short signk =-(k<<1) + 1;
    	short s;
	//stage1:
	
	short r0 = (g%2)*(g>>(n-1));//(g>>(n-1))*((g>>(n-2))%2);
	short signr0 = -(r0<<1) + 1;
	SB[ind0 + k*D + r0] = signr0*(SA[ind0] + signk*SA[ind1]);
	SB[ind0 + k*D +(!r0)] = SA[ind0+1] + signk*SA[ind1+1];

	//SB[ind0 + k*D ] = r0;
	//SB[ind0 + k*D +1] = r0;
	__syncthreads();
	A[j+blockIdx.x*(N>>1)] = SB[j+(!blockIdx.x)*(N>>1)];
	__syncthreads();
	SB[j+(!blockIdx.x)*(N>>1)] = A[j+(!blockIdx.x)*(N>>1)];
	__syncthreads();

	 //stage2:
	s=2;
	
	short r1 = (((g%2)<<1) + ((g>>(n-s+1))%2))*((g%(1<<(n-s+1)))>>1);

	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];
	//__syncthreads();
	SB[ind2] = SA[ind2]*SROT[r1<<1] + SA[ind2+1]*SROT[(r1<<1)+1];
	SB[ind2+1] = -SA[ind2]*SROT[(r1<<1)+1] + SA[ind2+1]*SROT[r1<<1];
	__syncthreads();
	//stage3:

	short r2= k*(j>>(n-2));
	short signr2 = -(r2<<1) + 1;
	SA[ind2 + r2] = signr2*(SB[ind0] + signk*SB[ind1]);
	SA[ind2 + (!r2)] = SB[ind0+1] + signk*SB[ind1+1];
	__syncthreads();
	//stage4:

	s=4;
	short r3 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>(s-1));
	SB[ind2] = SA[ind0] + signk*SA[ind1];
	SB[ind2+1] = SA[ind0+1] + signk*SA[ind1+1];
	SA[ind2] = SB[ind2]*SROT[r3<<1] + SB[ind2+1]*SROT[(r3<<1)+1];
	SA[ind2+1] = -SB[ind2]*SROT[(r3<<1)+1] + SB[ind2+1]*SROT[r3<<1];
	__syncthreads();
	//stage5:
	SB[ind2 + r2] = signr2*(SA[ind0] + (-(k*2)+1)*SA[ind1]);
	SB[ind2 + (!r2)] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();
	//stage6:

	s=6;
	short r5 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>(s-1));
	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];
	SB[ind2] = SA[ind2]*SROT[r5<<1] + SA[ind2+1]*SROT[(r5<<1)+1];
	SB[ind2+1] = -SA[ind2]*SROT[(r5<<1)+1] + SA[ind2+1]*SROT[r5<<1];
	__syncthreads();

	//stage7:
	SA[ind2 + r2] = signr2*(SB[ind0] + (-(k*2)+1)*SB[ind1]);
	SA[ind2 + (!r2)] = SB[ind0+1] + signk*SB[ind1+1];
	__syncthreads();
	//stage8:

	s=8;
	short r7 = (1<<(s-2))*(((j%2)<<1) + ((j>>1)%2))*(j>>(s-1));
	SB[ind2] = SA[ind0] + signk*SA[ind1];
	SB[ind2+1] = SA[ind0+1] + signk*SA[ind1+1];
	SA[ind2] = SB[ind2]*SROT[r7<<1] + SB[ind2+1]*SROT[(r7<<1)+1];
	SA[ind2+1] = -SB[ind2]*SROT[(r7<<1)+1] + SB[ind2+1]*SROT[r7<<1];
	__syncthreads();


	//stage9:
	SB[ind2 + r2] = signr2*(SA[ind0] + (-(k*2)+1)*SA[ind1]);
	SB[ind2 + (!r2)] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();
	//stage10:
	SA[ind2] = SB[ind0] + signk*SB[ind1];
	SA[ind2+1] = SB[ind0+1] + signk*SB[ind1+1];

	//SB[ind2] = SA[ind0] + signk*SA[ind1];
	//SB[ind2+1] = SA[ind0+1] + signk*SA[ind1+1];
	__syncthreads();

	A[j+blockIdx.x*(N)] = SA[j];
	A[j+blockDim.x+blockIdx.x*(N)] = SA[j+blockDim.x];

      }

       
#include <hip/hip_runtime.h>
#include  <stdio.h> 
       #include  <math.h>
       int  main() 
       { 
        
           float A[2*N]; 
           float *Ad;
	   float ROT[2*N];
	   float *ROTd; 
	  
           int memsize= 2*N * sizeof(float); 
	   int rotsize = 2*N* sizeof(float);


		for(int i=0; i<N; i++)
		{
			A[2*i]  = i;
			A[2*i+1]= i;	
		}
		for(int j=0; j < (N); j++)
		{
			
			ROT[2*j]= cosf((j*(6.2857))/N);
			ROT[2*j+1]=sinf((j*(6.2857))/N);	
		}

           hipMalloc((void**)&Ad, memsize);
	   hipMalloc((void**)&ROTd, rotsize); 

           hipMemcpy(Ad, A, memsize,  hipMemcpyHostToDevice); 
	   hipMemcpy(ROTd, ROT, rotsize,  hipMemcpyHostToDevice); 

           //__global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter); 
           dim3 gridDim(2,1);
           dim3 blockDim(D,1);
	   fft<<<gridDim , blockDim>>>(Ad,ROTd );
           hipMemcpy(A, Ad, memsize,  hipMemcpyDeviceToHost); 
	   hipMemcpy(ROT, ROTd, rotsize,  hipMemcpyDeviceToHost);

            printf("The  outputs are: \n");
            for (int l=0; l< N; l++) 
            printf("RE:A[%d]=%f\t\t\t, IM: A[%d]=%f\t\t\t \n ",2*l,A[2*l],2*l+1,A[2*l+1]); 

     }

